#include "hip/hip_runtime.h"
//
// Created by sriram on 5/13/25.
//

#include "api.h"
#include <hip/hip_cooperative_groups.h>
#include <cuda/barrier>
#include <cuda/pipeline>
#include <random>

using barrier = cuda::barrier<cuda::thread_scope_block>;

__device__ __host__ __forceinline__ constexpr size_t ceil_div(const size_t top, const size_t bottom) {
    return (top + (bottom - 1)) / bottom;
}

template<size_t M, size_t N>
__device__ void print_matrix(float matrix[M][N]) {
    if (blockIdx.x + blockIdx.y == 0 && threadIdx.x + threadIdx.y == 0) {
        printf("\n");
        for (size_t m{0}; m < M; ++m) {
            printf("[");
            for (size_t n{0}; n < N; ++n) {
                printf("%f ", matrix[m][n]);
            }
            printf("]\n");
        }
    }
}

#define CUDA_0_EXPR(expr)                                                \
{                                                                        \
    if (threadIdx.x + threadIdx.y == 0 && blockIdx.x + blockIdx.y == 0){ \
        expr                                                             \
    }                                                                    \
}

template<
    typename T,
    typename VECTOR_TYPE = int4,
    size_t BLOCK_TILE_SIZE_X,
    size_t BLOCK_TILE_SIZE_Y,
    size_t BLOCK_TILE_SIZE_K,
    size_t THREADS_PER_BLOCK,
    size_t BLOCK_TILE_SKEW_SIZE_X = 0,
    size_t BLOCK_TILE_SKEW_SIZE_Y = 0
>
__device__ void load_data_to_shared_memory_transposed_vectorized(
    const T *matrix_one,
    const T *matrix_two,
    const size_t stride_one,
    const size_t stride_two,
    T one_shared[BLOCK_TILE_SIZE_K][BLOCK_TILE_SIZE_Y + BLOCK_TILE_SKEW_SIZE_Y],
    T two_shared[BLOCK_TILE_SIZE_K][BLOCK_TILE_SIZE_X + BLOCK_TILE_SKEW_SIZE_X],
    const size_t mat_one_rows,
    const size_t mat_two_columns,
    const size_t shared,
    const uint iteration,
    const uint thread_linear_idx,
    VECTOR_TYPE v0
) {
    constexpr size_t units_per_vector{sizeof(VECTOR_TYPE) / sizeof(T)};
    static_assert(sizeof(VECTOR_TYPE) % sizeof(T) == 0);

    // ensure there will be an even amount of vectorized loads
    static_assert(BLOCK_TILE_SIZE_X % units_per_vector == 0);
    static_assert(BLOCK_TILE_SIZE_K % units_per_vector == 0);

#ifndef BENCHMARK
    // ensures leading dimensions are padded to handle additional reads
    assert(stride_one % units_per_vector == 0);
    assert(stride_two % units_per_vector == 0);
#endif

    // We need to make sure the data alignment is correct.
    static_assert((BLOCK_TILE_SIZE_Y) * sizeof(T) % sizeof(VECTOR_TYPE) == 0U);
    static_assert((BLOCK_TILE_SIZE_X) * sizeof(T) % sizeof(VECTOR_TYPE) == 0U);

    static_assert((BLOCK_TILE_SIZE_Y + BLOCK_TILE_SKEW_SIZE_Y) * sizeof(T) % sizeof(VECTOR_TYPE) == 0U);
    static_assert((BLOCK_TILE_SIZE_X + BLOCK_TILE_SKEW_SIZE_X) * sizeof(T) % sizeof(VECTOR_TYPE) == 0U);

    // scaling the load number down to account for the vectorized size
    constexpr size_t VEC_BLOCK_TILE_SIZE_X{BLOCK_TILE_SIZE_X / units_per_vector};
    constexpr size_t VEC_BLOCK_TILE_SIZE_K{BLOCK_TILE_SIZE_K / units_per_vector};

    // determines how many vectorized loads are performed per thread
    constexpr size_t one_iterations{
        ceil_div(BLOCK_TILE_SIZE_Y * VEC_BLOCK_TILE_SIZE_K, THREADS_PER_BLOCK)
    };

    // load into matrix one
#pragma unroll
    for (size_t one_iter{0}; one_iter < one_iterations; ++one_iter) {
        const size_t one_shared_row{(thread_linear_idx + one_iter * THREADS_PER_BLOCK) / VEC_BLOCK_TILE_SIZE_K};
        const size_t one_shared_column{
            (thread_linear_idx + one_iter * THREADS_PER_BLOCK) % VEC_BLOCK_TILE_SIZE_K * units_per_vector
        };

        const size_t mat_one_row{blockIdx.y * BLOCK_TILE_SIZE_Y + one_shared_row};
        const size_t mat_one_column{iteration * BLOCK_TILE_SIZE_K + one_shared_column};

        VECTOR_TYPE mat_one_row_vector_vals{v0};

        // if in bounds we save the data to the temp register value mat_one_row_vector_vals
        if (mat_one_row < mat_one_rows && mat_one_column < shared) {
            const VECTOR_TYPE *mat_one_vec_ptr{
                reinterpret_cast<const VECTOR_TYPE *>(matrix_one + (mat_one_row * stride_one) + mat_one_column)
            };
            mat_one_row_vector_vals = *mat_one_vec_ptr;
        }

        // Transposed store of the data back into shared memory
        if (one_shared_row < BLOCK_TILE_SIZE_Y && one_shared_column < BLOCK_TILE_SIZE_K) {
            for (size_t i{0}; i < units_per_vector; ++i) {
                one_shared[one_shared_column + i][one_shared_row] =
                        reinterpret_cast<const T *>(&mat_one_row_vector_vals)[i];
            }
        }
    }

    constexpr size_t two_iterations{ceil_div(BLOCK_TILE_SIZE_K * VEC_BLOCK_TILE_SIZE_X, THREADS_PER_BLOCK)};

    // load into matrix two
#pragma unroll
    for (size_t two_iter{0}; two_iter < two_iterations; ++two_iter) {
        const size_t two_shared_row{(thread_linear_idx + two_iter * THREADS_PER_BLOCK) / VEC_BLOCK_TILE_SIZE_X};

        const size_t two_shared_column{
            (thread_linear_idx + two_iter * THREADS_PER_BLOCK) % VEC_BLOCK_TILE_SIZE_X * units_per_vector
        };

        const size_t mat_two_row{iteration * BLOCK_TILE_SIZE_K + two_shared_row};
        const size_t mat_two_column{blockIdx.x * BLOCK_TILE_SIZE_X + two_shared_column};

        VECTOR_TYPE mat_two_row_vector_vals{v0};

        // if in bounds we save the data to the temp register value mat_two_row_vector_vals
        if (mat_two_row < shared && mat_two_column < mat_two_columns) {
            const VECTOR_TYPE *mat_two_vec_ptr{
                reinterpret_cast<const VECTOR_TYPE *>(matrix_two + (mat_two_row * stride_two) + mat_two_column)
            };

            mat_two_row_vector_vals = *mat_two_vec_ptr;
        }

        if (two_shared_row < BLOCK_TILE_SIZE_K && two_shared_column < BLOCK_TILE_SIZE_X) {
            *reinterpret_cast<VECTOR_TYPE *>(&two_shared[two_shared_row][two_shared_column]) =
                    mat_two_row_vector_vals;
        }
    }
}

// TODO see if vectorized copies are better (16 bytes at at time)
// only possible for B matrices
template<
    typename T,
    size_t BLOCK_TILE_SIZE_X,
    size_t BLOCK_TILE_SIZE_K,
    size_t THREADS_PER_BLOCK,
    size_t COPY_ITERATIONS,
    size_t BLOCK_TILE_SKEW_X = 0
>
__device__ __forceinline__ void load_data_to_shared_matrix_B_async(
    T B_shared[BLOCK_TILE_SIZE_K][BLOCK_TILE_SIZE_X + BLOCK_TILE_SKEW_X],
    const T *B_matrix,
    const size_t k,
    const size_t n,
    const size_t leading_dimension,
    const uint iteration,
    const uint thread_linear_idx,
    cuda::pipeline<cuda::thread_scope_thread> &B_shared_pipeline
) {
    // ensures even amount of copies per thread
    static_assert(BLOCK_TILE_SIZE_K * BLOCK_TILE_SIZE_X % THREADS_PER_BLOCK == 0);

    // determines how many copies are performed per thread
    constexpr size_t copy_iterations{
        ceil_div(BLOCK_TILE_SIZE_K * BLOCK_TILE_SIZE_X, THREADS_PER_BLOCK)
    };

    static_assert(copy_iterations == COPY_ITERATIONS);

#pragma unroll
    for (size_t copy_iter{0}; copy_iter < COPY_ITERATIONS; ++copy_iter) {
        const size_t shared_row{
            (thread_linear_idx + copy_iter * THREADS_PER_BLOCK) / BLOCK_TILE_SIZE_X
        };

        const size_t shared_column{
            (thread_linear_idx + copy_iter * THREADS_PER_BLOCK) % BLOCK_TILE_SIZE_X
        };

        const size_t B_row{iteration * BLOCK_TILE_SIZE_K + shared_row};
        const size_t B_column{blockIdx.x * BLOCK_TILE_SIZE_X + shared_column};

        T value{static_cast<T>(0)};
        const T *p_value;

        if (B_row < k && B_column < n)
            p_value = B_matrix + (leading_dimension * B_row + B_column);
        else
            p_value = &value;

        B_shared_pipeline.producer_acquire();

        cuda::memcpy_async(
            &B_shared[shared_row][shared_column],
            p_value,
            sizeof(T),
            B_shared_pipeline
        );

        B_shared_pipeline.producer_commit();
    }
}


template<
    typename T,
    size_t BLOCK_TILE_SIZE_Y,
    size_t BLOCK_TILE_SIZE_K,
    size_t THREADS_PER_BLOCK,
    size_t COPY_ITERATIONS,
    size_t BLOCK_TILE_SKEW_K = 0
>
__device__ __forceinline__ void load_data_to_shared_matrix_A_async(
    T A_shared[BLOCK_TILE_SIZE_Y][BLOCK_TILE_SIZE_K + BLOCK_TILE_SKEW_K],
    const T *A_matrix,
    const size_t m,
    const size_t k,
    const size_t leading_dimension,
    const uint iteration,
    const uint thread_linear_idx,
    cuda::pipeline<cuda::thread_scope_thread> &A_shared_pipeline
) {
    // ensures even amount of copies per thread
    static_assert(BLOCK_TILE_SIZE_K * BLOCK_TILE_SIZE_Y % THREADS_PER_BLOCK == 0);

    // determines how many copies are performed per thread
    constexpr size_t copy_iterations{
        ceil_div(BLOCK_TILE_SIZE_K * BLOCK_TILE_SIZE_Y, THREADS_PER_BLOCK)
    };

    static_assert(copy_iterations == COPY_ITERATIONS);

#pragma unroll
    for (size_t copy_iter{0}; copy_iter < COPY_ITERATIONS; ++copy_iter) {
        const size_t shared_row{
            (thread_linear_idx + copy_iter * THREADS_PER_BLOCK) / BLOCK_TILE_SIZE_K
        };

        const size_t shared_column{
            (thread_linear_idx + copy_iter * THREADS_PER_BLOCK) % BLOCK_TILE_SIZE_K
        };

        const size_t A_row{BLOCK_TILE_SIZE_Y * blockIdx.y + shared_row};
        const size_t A_column{iteration * BLOCK_TILE_SIZE_K + shared_column};

        T value{static_cast<T>(0)};
        const T *p_value;

        if (A_row < m && A_column < k)
            p_value = A_matrix + (leading_dimension * A_row + A_column);
        else
            p_value = &value;

        A_shared_pipeline.producer_acquire();

        cuda::memcpy_async(
            &A_shared[shared_row][shared_column],
            p_value,
            sizeof(T),
            A_shared_pipeline
        );

        A_shared_pipeline.producer_commit();
    }
}

template<
    typename T,
    size_t BLOCK_TILE_SIZE_Y,
    size_t BLOCK_TILE_SIZE_K,
    size_t THREADS_PER_BLOCK,
    size_t COPY_ITERATIONS,
    size_t BLOCK_TILE_SKEW_Y = 0
>
__device__ __forceinline__ void load_data_to_shared_matrix_A_transposed_async(
    T A_shared_T[BLOCK_TILE_SIZE_K][BLOCK_TILE_SIZE_Y + BLOCK_TILE_SKEW_Y],
    const T *A_matrix,
    const size_t k,
    const size_t m,
    const size_t leading_dimension,
    const uint iteration,
    const uint thread_linear_idx,
    cuda::pipeline<cuda::thread_scope_thread> &A_shared_pipeline
) {
    // ensures even amount of copies per thread
    static_assert(BLOCK_TILE_SIZE_K * BLOCK_TILE_SIZE_Y % THREADS_PER_BLOCK == 0);

    // determines how many copies are performed per thread
    constexpr size_t copy_iterations{
        ceil_div(BLOCK_TILE_SIZE_K * BLOCK_TILE_SIZE_Y, THREADS_PER_BLOCK)
    };

    static_assert(copy_iterations == COPY_ITERATIONS);

#pragma unroll
    for (size_t copy_iter{0}; copy_iter < copy_iterations; ++copy_iter) {
        const size_t shared_row{
            (thread_linear_idx + copy_iter * THREADS_PER_BLOCK) / BLOCK_TILE_SIZE_K
        };

        const size_t shared_column{
            (thread_linear_idx + copy_iter * THREADS_PER_BLOCK) % BLOCK_TILE_SIZE_K
        };

        const size_t A_row{blockIdx.y * BLOCK_TILE_SIZE_Y + shared_row};
        const size_t A_column{iteration * BLOCK_TILE_SIZE_K + shared_column};

        T value{static_cast<T>(0)};
        const T *p_value;

        if (A_row < m && A_column < k)
            p_value = A_matrix + (leading_dimension * A_row + A_column);
        else
            p_value = &value;

        A_shared_pipeline.producer_acquire();

        cuda::memcpy_async(
            &A_shared_T[shared_column][shared_row],
            p_value,
            sizeof(T),
            A_shared_pipeline
        );

        A_shared_pipeline.producer_commit();
    }
}

template<
    bool TRANSPOSED_A,
    size_t BLOCK_TILE_SIZE_Y,
    size_t BLOCK_TILE_SIZE_K,
    size_t Y_SKEW,
    size_t K_SKEW,
    bool IS_ROW>
constexpr uint A_shared_dim() {
    if constexpr (TRANSPOSED_A) {
        if constexpr (IS_ROW)
            return BLOCK_TILE_SIZE_K + K_SKEW;
        else
            return BLOCK_TILE_SIZE_Y + Y_SKEW;
    } else {
        if constexpr (IS_ROW)
            return BLOCK_TILE_SIZE_Y + Y_SKEW;
        else return BLOCK_TILE_SIZE_K + K_SKEW;
    }
}

template<
    typename T,
    size_t BLOCK_TILE_SIZE_X,
    size_t BLOCK_TILE_SIZE_Y,
    size_t BLOCK_TILE_SIZE_K,
    size_t THREADS_PER_BLOCK,
    size_t COPY_ITERATIONS_A,
    size_t COPY_ITERATIONS_B,
    size_t BLOCK_TILE_SKEW_X = 0,
    size_t BLOCK_TILE_SKEW_Y = 0,
    size_t BLOCK_TILE_SKEW_K = 0,
    bool TRANSPOSE_A = true
>
__device__ void load_data_to_shared_async(
    T A_shared[A_shared_dim<
        TRANSPOSE_A,
        BLOCK_TILE_SIZE_Y,
        BLOCK_TILE_SIZE_K,
        BLOCK_TILE_SKEW_Y,
        BLOCK_TILE_SKEW_K,
        true>()]
    [A_shared_dim<
        TRANSPOSE_A,
        BLOCK_TILE_SIZE_Y,
        BLOCK_TILE_SIZE_K,
        BLOCK_TILE_SKEW_Y,
        BLOCK_TILE_SKEW_K,
        false>()],
    T B_shared[BLOCK_TILE_SIZE_K][BLOCK_TILE_SIZE_X + BLOCK_TILE_SKEW_X],
    const T *A_matrix,
    const T *B_matrix,
    const size_t k,
    const size_t m,
    const size_t n,
    const size_t leading_dimension_A,
    const size_t leading_dimension_B,
    const uint iteration,
    const uint thread_linear_idx,
    cuda::pipeline<cuda::thread_scope_thread> &A_shared_pipeline,
    cuda::pipeline<cuda::thread_scope_thread> &B_shared_pipeline
) {
    if constexpr (TRANSPOSE_A) {
        load_data_to_shared_matrix_A_transposed_async<
            T, BLOCK_TILE_SIZE_Y, BLOCK_TILE_SIZE_K,
            THREADS_PER_BLOCK, COPY_ITERATIONS_A,
            BLOCK_TILE_SKEW_Y
        >(
            A_shared,
            A_matrix,
            k,
            m,
            leading_dimension_A,
            iteration,
            thread_linear_idx,
            A_shared_pipeline
        );
    } else {
        load_data_to_shared_matrix_A_async<
            T, BLOCK_TILE_SIZE_Y, BLOCK_TILE_SIZE_K,
            THREADS_PER_BLOCK, COPY_ITERATIONS_A,
            BLOCK_TILE_SKEW_Y
        >(
            A_shared,
            A_matrix,
            m,
            k,
            leading_dimension_A,
            iteration,
            thread_linear_idx,
            A_shared_pipeline
        );
    }

    load_data_to_shared_matrix_B_async<
        T, BLOCK_TILE_SIZE_X, BLOCK_TILE_SIZE_K,
        THREADS_PER_BLOCK, COPY_ITERATIONS_B,
        BLOCK_TILE_SKEW_X
    >(
        B_shared,
        B_matrix,
        k,
        n,
        leading_dimension_B,
        iteration,
        thread_linear_idx,
        B_shared_pipeline
    );
}

template<
    typename T,
    size_t BLOCK_TILE_SIZE_X,
    size_t BLOCK_TILE_SIZE_Y,
    size_t BLOCK_TILE_SIZE_K,
    size_t WARP_TILE_SIZE_X,
    size_t WARP_TILE_SIZE_Y,
    size_t THREAD_TILE_SIZE_X,
    size_t THREAD_TILE_SIZE_Y,
    size_t NUM_THREADS_PER_WARP_X,
    size_t NUM_THREADS_PER_WARP_Y
>
__global__ void gemm_2DBT_2DWT_2DTT_vload(
    const T *matrix_one,
    const T *matrix_two,
    T *matrix_dest,
    const T alpha,
    const T beta,
    const size_t mat_one_rows,
    const size_t mat_two_columns,
    const size_t shared,
    const size_t row_stride_one,
    const size_t row_stride_two,
    const size_t row_stride_dest) {
    __shared__ T mat_one_thread_block_tile_transposed[BLOCK_TILE_SIZE_K][BLOCK_TILE_SIZE_Y];
    __shared__ T mat_two_thread_block_tile[BLOCK_TILE_SIZE_K][BLOCK_TILE_SIZE_X];

    // One Warp TILE will be of size WARP_TILE_SIZE_X x WARP_TILE_SIZE_Y
    // One Warp will be responsible for each Warp block, ideally multiple warp blocks
    // will be able to fit in one regular block allowing multiple warps to exist per
    // block

    // EACH block computes BLOCK_TILE_SIZE_X * BLOCK_TILE_SIZE_Y outputs of C

    // max threads per warp is 32, so we ensure that the warp block also complies
    // with this.
    static_assert(NUM_THREADS_PER_WARP_X * NUM_THREADS_PER_WARP_Y == 32);

    // We need to figure out a couple of things,
    // 1) we need to figure out how many WARP Tiles will be present in
    // the x and y direction similar as to calculating how many blocks will
    // be in the grid for a GPU launch we are doing the same but making a block
    // the grid and having our WARP TILE Be the new block
    //
    // 2) This is needed to calculate the total amount of THREADS per block in a
    // constant way
    constexpr size_t NUM_WARPS_PER_BLOCK_X{BLOCK_TILE_SIZE_X / WARP_TILE_SIZE_X};
    static_assert(BLOCK_TILE_SIZE_X % WARP_TILE_SIZE_X == 0);

    // repeat for y dimension
    constexpr size_t NUM_WARPS_PER_BLOCK_Y{BLOCK_TILE_SIZE_Y / WARP_TILE_SIZE_Y};
    static_assert(BLOCK_TILE_SIZE_Y % WARP_TILE_SIZE_Y == 0);

    // so total amount of warp tiles in a block would be
    // NUM_WARPS_PER_BLOCK_X * NUM_WARPS_PER_BLOCK_Y

    // In the previous implementation each thread had 2 register caches
    //
    // one cache cached several values in the y dimension from matrix one, total
    // elements are THREAD_TILE_SIZE_Y
    //
    // two cache cached several values in the x dimension from matrix two, total
    // elements are THREAD_TILE_SIZE_X
    //
    // In the end these values were reused for multiplication computing a total
    // of THREAD_TILE_SIZE_Y x THREAD_TILE_SIZE_X elements.
    //
    // Now that we are bounding warps to WARP_TILE sizes the amount of data being
    // computed by each thread may go up. So we know need to adjust the amount
    // of values being cached and computed to reflect this.
    //
    // We ideally want to keep our thread tile sizes consistent so we
    // instead add an extra dimension to each cache
    constexpr size_t NUM_CACHES_PER_WARP_X{
        WARP_TILE_SIZE_X / (THREAD_TILE_SIZE_X * NUM_THREADS_PER_WARP_X)
    };

    // repeat for y TILE cache
    constexpr size_t NUM_CACHES_PER_WARP_Y{
        WARP_TILE_SIZE_Y / (THREAD_TILE_SIZE_Y * NUM_THREADS_PER_WARP_Y)
    };

    static_assert(WARP_TILE_SIZE_X % (THREAD_TILE_SIZE_X * NUM_THREADS_PER_WARP_X) == 0);
    static_assert(WARP_TILE_SIZE_Y % (THREAD_TILE_SIZE_Y * NUM_THREADS_PER_WARP_Y) == 0);

    // Now we create the caches with the extra dimension
    T one_cache[NUM_CACHES_PER_WARP_Y][THREAD_TILE_SIZE_Y] = {static_cast<T>(0)};
    T two_cache[NUM_CACHES_PER_WARP_X][THREAD_TILE_SIZE_X] = {static_cast<T>(0)};

    // since we have more caches we will have more intermediates (values computed per thread)
    // as well, so we add extra dimensions here as well reflecting this
    T intermediates[NUM_CACHES_PER_WARP_Y][NUM_CACHES_PER_WARP_X][THREAD_TILE_SIZE_Y][
        THREAD_TILE_SIZE_X] = {static_cast<T>(0)};

    // now we can also easily calculate the total threads per block, needed for loading data
    constexpr size_t THREADS_PER_BLOCK{NUM_WARPS_PER_BLOCK_X * NUM_WARPS_PER_BLOCK_Y * 32};

    // this kernel should be launched with a 1d block so the linear dimension is just the threadidx.x
    const size_t thread_linear_idx{threadIdx.x};

    // the linear idx of the warp in the thread block
    const size_t warp_linear_idx{thread_linear_idx / warpSize};

    // Now lets figure out what warp that linear idx maps too (x, y)
    const size_t warp_row_idx{warp_linear_idx / NUM_WARPS_PER_BLOCK_X};
    const size_t warp_col_idx{warp_linear_idx % NUM_WARPS_PER_BLOCK_X};

    // figure out what row and column we are in the warp
    const size_t thread_linear_idx_in_warp{thread_linear_idx % warpSize};
    const size_t thread_idx_in_warp_row{thread_linear_idx_in_warp / NUM_THREADS_PER_WARP_X};
    const size_t thread_idx_in_warp_column{thread_linear_idx_in_warp % NUM_THREADS_PER_WARP_X};

    constexpr size_t units_per_vector{sizeof(int4) / sizeof(T)};

    // ensure int4 can be event split up by the base TYPE necessary for conversion
    static_assert(sizeof(int4) % sizeof(T) == 0);

    // we will store data along these dimensions for vectorized storage they need to be divisible
    static_assert(BLOCK_TILE_SIZE_K % units_per_vector == 0);
    static_assert(BLOCK_TILE_SIZE_X % units_per_vector == 0);

    static_assert(THREAD_TILE_SIZE_X % units_per_vector == 0);
    static_assert(THREAD_TILE_SIZE_Y % units_per_vector == 0);

    // This determines how many vectorized loads we need to perform to fill one tile
    constexpr size_t vectorized_thread_tile_size_x{THREAD_TILE_SIZE_X / units_per_vector};
    constexpr size_t vectorized_thread_tile_size_y{THREAD_TILE_SIZE_Y / units_per_vector};

    const size_t total_iters{ceil_div(shared, BLOCK_TILE_SIZE_K)};

    for (size_t iter{0}; iter < total_iters; ++iter) {
        load_data_to_shared_memory_transposed_vectorized<
            T, int4,
            BLOCK_TILE_SIZE_X,
            BLOCK_TILE_SIZE_Y,
            BLOCK_TILE_SIZE_K,
            THREADS_PER_BLOCK
        >(
            matrix_one,
            matrix_two,
            row_stride_one,
            row_stride_two,
            mat_one_thread_block_tile_transposed,
            mat_two_thread_block_tile,
            mat_one_rows,
            mat_two_columns,
            shared,
            iter,
            thread_linear_idx,
            int4{0, 0, 0, 0}
        );

        __syncthreads();

        // #pragma unroll
        for (size_t k{0}; k < BLOCK_TILE_SIZE_K; ++k) {
            // we need to start filling the one matrix cache
#pragma unroll
            for (size_t y_cache_idx{0}; y_cache_idx < NUM_CACHES_PER_WARP_Y; ++y_cache_idx) {
                // Here we calculate the row in the shared block based on the warp coordinates
                // and the thread coordinates

                // To calculate the row we first multiply the warp block y coordinate by the
                // Warp y dimension scale on the grid scale this is equivalent to doing blockIdx.y * blockDim.y
                // Next based on what cache we are in we need to skip that many rows. We do this by multiplying the
                // y_cache_idx by (WARP_TILE_SIZE_Y / NUM_CACHES_PER_WARP_Y) this value is equivalent too
                // (THREAD_TILE_SIZE_Y * NUM_THREADS_PER_WARP_Y) which is how many distinct rows are processed per
                // one cache fill of warp tile. Finally, we add the row that this thread is part of in the warp.

                // each thread loads TILE_SIZE_Y
                // assuming this configuration NUM_THREADS_PER_WARP_X = 4, and NUM_THREADS_PER_WARP_Y = 8
                // we can assume this load pattern
                // Threads [0 to 3] load rows [0 to 7], Threads [4 to 7] load rows [8 to 15] ...
                // Threads [28 to 31] load rows [54 to 63], this would result in a bank conflict for each
                // new warp_row and a broadcast for all threads in warp row, but luckily
                // the shared memory is transposed resulting in only broadcasts
                const size_t one_shared_row_idx{
                    warp_row_idx * WARP_TILE_SIZE_Y +
                    y_cache_idx * (WARP_TILE_SIZE_Y / NUM_CACHES_PER_WARP_Y) +
                    thread_idx_in_warp_row * THREAD_TILE_SIZE_Y
                };

                const auto one_shared_ptr{
                    reinterpret_cast<int4 *>(&mat_one_thread_block_tile_transposed[k][one_shared_row_idx])
                };

                auto tile_ptr{
                    reinterpret_cast<int4 *>(&one_cache[y_cache_idx])
                };

                // load into register cache one[y_cache_idx] with vectorized loads
                // #pragma unroll
                for (size_t vy_iter{0}; vy_iter < vectorized_thread_tile_size_y; ++vy_iter)
                    tile_ptr[vy_iter] = one_shared_ptr[vy_iter];
            }

#pragma unroll
            for (size_t x_cache_id{0}; x_cache_id < NUM_CACHES_PER_WARP_X; ++x_cache_id) {
                const size_t two_shared_col_idx{
                    warp_col_idx * WARP_TILE_SIZE_X +
                    x_cache_id * (WARP_TILE_SIZE_X / NUM_CACHES_PER_WARP_X) +
                    thread_idx_in_warp_column * THREAD_TILE_SIZE_X
                };

                const auto two_shared_ptr{
                    reinterpret_cast<int4 *>(&mat_two_thread_block_tile[k][two_shared_col_idx])
                };

                auto tile_ptr{
                    reinterpret_cast<int4 *>(&two_cache[x_cache_id])
                };

                // #pragma unroll
                for (size_t vx_iter{0}; vx_iter < vectorized_thread_tile_size_x; ++vx_iter)
                    tile_ptr[vx_iter] = two_shared_ptr[vx_iter];
            }

            // compute intermediates
#pragma unroll
            for (size_t y_cache_idx{0}; y_cache_idx < NUM_CACHES_PER_WARP_Y; ++y_cache_idx) {
                // #pragma unroll
                for (size_t x_cache_idx{0}; x_cache_idx < NUM_CACHES_PER_WARP_X; ++x_cache_idx) {
#pragma unroll
                    for (size_t one_cache_idx{0}; one_cache_idx < THREAD_TILE_SIZE_Y; ++one_cache_idx) {
                        T one_cache_value{one_cache[y_cache_idx][one_cache_idx]};
                        // #pragma unroll
                        for (size_t two_cache_index{0}; two_cache_index < THREAD_TILE_SIZE_X; ++two_cache_index) {
                            intermediates[y_cache_idx][x_cache_idx][one_cache_idx][two_cache_index] +=
                                    one_cache_value * two_cache[x_cache_idx][two_cache_index];
                        }
                    }
                }
            }
        }
        __syncthreads();
    }

    // vectorized store back into the dest matrix
#pragma unroll
    for (size_t y_cache_idx{0}; y_cache_idx < NUM_CACHES_PER_WARP_Y; ++y_cache_idx) {
#pragma unroll
        for (size_t x_cache_idx{0}; x_cache_idx < NUM_CACHES_PER_WARP_X; ++x_cache_idx) {
            // #pragma unroll
            for (size_t one_cache_idx{0}; one_cache_idx < THREAD_TILE_SIZE_Y; ++one_cache_idx) {
                const size_t dest_row{
                    BLOCK_TILE_SIZE_Y * blockIdx.y +
                    warp_row_idx * WARP_TILE_SIZE_Y +
                    y_cache_idx * (WARP_TILE_SIZE_Y / NUM_CACHES_PER_WARP_Y) +
                    thread_idx_in_warp_row * THREAD_TILE_SIZE_Y + one_cache_idx
                };

                const size_t dest_column{
                    BLOCK_TILE_SIZE_X * blockIdx.x +
                    warp_col_idx * WARP_TILE_SIZE_X +
                    x_cache_idx * (WARP_TILE_SIZE_X / NUM_CACHES_PER_WARP_X) +
                    thread_idx_in_warp_column * THREAD_TILE_SIZE_X
                };

                auto dest_ptr{&matrix_dest[dest_row * row_stride_dest + dest_column]};
                T *tile_ptr{&intermediates[y_cache_idx][x_cache_idx][one_cache_idx][0]};

                // #pragma unroll
                for (size_t two_cache_vec_idx{0}; two_cache_vec_idx < vectorized_thread_tile_size_x; ++
                     two_cache_vec_idx) {
                    if (dest_row < mat_one_rows && (
                            dest_column + two_cache_vec_idx * units_per_vector < mat_two_columns)) {
                        // #pragma unroll
                        for (size_t tile_idx{0}; tile_idx < units_per_vector; ++tile_idx) {
                            tile_ptr[tile_idx] = tile_ptr[tile_idx] * alpha + dest_ptr[tile_idx] * beta;
                        }

                        reinterpret_cast<int4 *>(dest_ptr)[two_cache_vec_idx] =
                                reinterpret_cast<int4 *>(tile_ptr)[two_cache_vec_idx];
                    }
                }
            }
        }
    }
}

template<
    typename T,
    size_t BLOCK_TILE_SIZE_X,
    size_t BLOCK_TILE_SIZE_Y,
    size_t BLOCK_TILE_SIZE_K,
    size_t WARP_TILE_SIZE_X,
    size_t WARP_TILE_SIZE_Y,
    size_t THREAD_TILE_SIZE_X,
    size_t THREAD_TILE_SIZE_Y,
    size_t NUM_THREADS_PER_WARP_X,
    size_t NUM_THREADS_PER_WARP_Y,
    size_t STAGES = 2
>
__global__ void gemm_2DBT_2DWT_2DTT_async_load(
    const T *matrix_A,
    const T *matrix_B,
    T *matrix_C,
    const T alpha,
    const T beta,
    const size_t m,
    const size_t n,
    const size_t k,
    const size_t leading_dim_A,
    const size_t leading_dim_B,
    const size_t leading_dim_C) {
    // two buffer present compute loading overlap
    __shared__ T shared_A_T[STAGES][BLOCK_TILE_SIZE_K][BLOCK_TILE_SIZE_Y];
    __shared__ T shared_B[STAGES][BLOCK_TILE_SIZE_K][BLOCK_TILE_SIZE_X];

    // One Warp TILE will be of size WARP_TILE_SIZE_X x WARP_TILE_SIZE_Y
    // One Warp will be responsible for each Warp block, ideally multiple warp blocks
    // will be able to fit in one regular block allowing multiple warps to exist per
    // block

    // EACH block computes BLOCK_TILE_SIZE_X * BLOCK_TILE_SIZE_Y outputs of C

    // max threads per warp is 32, so we ensure that the warp block also complies
    // with this.
    static_assert(NUM_THREADS_PER_WARP_X * NUM_THREADS_PER_WARP_Y == 32);

    // We need to figure out a couple of things,
    // 1) we need to figure out how many WARP Tiles will be present in
    // the x and y direction similar as to calculating how many blocks will
    // be in the grid for a GPU launch we are doing the same but making a block
    // the grid and having our WARP TILE Be the new block
    //
    // 2) This is needed to calculate the total amount of THREADS per block in a
    // constant way
    constexpr size_t NUM_WARPS_PER_BLOCK_X{BLOCK_TILE_SIZE_X / WARP_TILE_SIZE_X};
    static_assert(BLOCK_TILE_SIZE_X % WARP_TILE_SIZE_X == 0);

    // repeat for y dimension
    constexpr size_t NUM_WARPS_PER_BLOCK_Y{BLOCK_TILE_SIZE_Y / WARP_TILE_SIZE_Y};
    static_assert(BLOCK_TILE_SIZE_Y % WARP_TILE_SIZE_Y == 0);

    // so total amount of warp tiles in a block would be
    // NUM_WARPS_PER_BLOCK_X * NUM_WARPS_PER_BLOCK_Y

    // In the previous implementation each thread had 2 register caches
    //
    // one cache cached several values in the y dimension from matrix one, total
    // elements are THREAD_TILE_SIZE_Y
    //
    // two cache cached several values in the x dimension from matrix two, total
    // elements are THREAD_TILE_SIZE_X
    //
    // In the end these values were reused for multiplication computing a total
    // of THREAD_TILE_SIZE_Y x THREAD_TILE_SIZE_X elements.
    //
    // Now that we are bounding warps to WARP_TILE sizes the amount of data being
    // computed by each thread may go up. So we know need to adjust the amount
    // of values being cached and computed to reflect this.
    //
    // We ideally want to keep our thread tile sizes consistent so we
    // instead add an extra dimension to each cache
    constexpr size_t NUM_CACHES_PER_WARP_X{
        WARP_TILE_SIZE_X / (THREAD_TILE_SIZE_X * NUM_THREADS_PER_WARP_X)
    };

    // repeat for y TILE cache
    constexpr size_t NUM_CACHES_PER_WARP_Y{
        WARP_TILE_SIZE_Y / (THREAD_TILE_SIZE_Y * NUM_THREADS_PER_WARP_Y)
    };

    static_assert(WARP_TILE_SIZE_X % (THREAD_TILE_SIZE_X * NUM_THREADS_PER_WARP_X) == 0);
    static_assert(WARP_TILE_SIZE_Y % (THREAD_TILE_SIZE_Y * NUM_THREADS_PER_WARP_Y) == 0);

    // Now we create the caches with the extra dimension
    T one_cache[NUM_CACHES_PER_WARP_Y][THREAD_TILE_SIZE_Y] = {static_cast<T>(0)};
    T two_cache[NUM_CACHES_PER_WARP_X][THREAD_TILE_SIZE_X] = {static_cast<T>(0)};

    // since we have more caches we will have more intermediates (values computed per thread)
    // as well, so we add extra dimensions here as well reflecting this
    T intermediates[NUM_CACHES_PER_WARP_Y][NUM_CACHES_PER_WARP_X][THREAD_TILE_SIZE_Y][
        THREAD_TILE_SIZE_X] = {static_cast<T>(0)};

    // now we can also easily calculate the total threads per block, needed for loading data
    constexpr size_t THREADS_PER_BLOCK{NUM_WARPS_PER_BLOCK_X * NUM_WARPS_PER_BLOCK_Y * 32};

    // this kernel should be launched with a 1d block so the linear dimension is just the threadidx.x
    const size_t thread_linear_idx{threadIdx.x};

    // the linear idx of the warp in the thread block
    const size_t warp_linear_idx{thread_linear_idx / 32};

    // Now lets figure out what warp that linear idx maps too (x, y)
    const size_t warp_row_idx{warp_linear_idx / NUM_WARPS_PER_BLOCK_X};
    const size_t warp_col_idx{warp_linear_idx % NUM_WARPS_PER_BLOCK_X};

    // figure out what row and column we are in the warp
    const size_t thread_linear_idx_in_warp{thread_linear_idx % warpSize};
    const size_t thread_idx_in_warp_row{thread_linear_idx_in_warp / NUM_THREADS_PER_WARP_X};
    const size_t thread_idx_in_warp_column{thread_linear_idx_in_warp % NUM_THREADS_PER_WARP_X};

    constexpr size_t units_per_vector{sizeof(int4) / sizeof(T)};

    // ensure int4 can be event split up by the base TYPE necessary for conversion
    static_assert(sizeof(int4) % sizeof(T) == 0);

    // we will store data along these dimensions for vectorized storage they need to be divisible
    static_assert(BLOCK_TILE_SIZE_K % units_per_vector == 0);
    static_assert(BLOCK_TILE_SIZE_X % units_per_vector == 0);

    static_assert(THREAD_TILE_SIZE_X % units_per_vector == 0);
    static_assert(THREAD_TILE_SIZE_Y % units_per_vector == 0);

    // This determines how many vectorized loads we need to perform to fill one tile
    constexpr size_t vectorized_thread_tile_size_x{THREAD_TILE_SIZE_X / units_per_vector};
    constexpr size_t vectorized_thread_tile_size_y{THREAD_TILE_SIZE_Y / units_per_vector};

    const size_t total_iters{ceil_div(k, BLOCK_TILE_SIZE_K)};

    // calculate acquires per load to shared
    constexpr size_t A_priors{BLOCK_TILE_SIZE_Y * BLOCK_TILE_SIZE_K / THREADS_PER_BLOCK};
    constexpr size_t B_priors{BLOCK_TILE_SIZE_X * BLOCK_TILE_SIZE_K / THREADS_PER_BLOCK};

    cuda::pipeline<cuda::thread_scope_thread> A_shared_pipeline{cuda::make_pipeline()};
    cuda::pipeline<cuda::thread_scope_thread> B_shared_pipeline{cuda::make_pipeline()};

    // preload both buffers
    for (size_t stage{0}; stage < STAGES; ++stage) {
        load_data_to_shared_async<
            T, BLOCK_TILE_SIZE_X,
            BLOCK_TILE_SIZE_Y, BLOCK_TILE_SIZE_K,
            THREADS_PER_BLOCK, A_priors, B_priors
        >(
            shared_A_T[stage],
            shared_B[stage],
            matrix_A,
            matrix_B,
            k,
            m,
            n,
            leading_dim_A,
            leading_dim_B,
            stage,
            thread_linear_idx,
            A_shared_pipeline,
            B_shared_pipeline
        );
    }

    size_t stage{0};

    // CUDA_0_EXPR(
    //     printf("total iters %lld", total_iters);
    // );

    for (size_t iter{0}; iter < total_iters; ++iter) {
        cuda::pipeline_consumer_wait_prior<A_priors>(A_shared_pipeline);
        cuda::pipeline_consumer_wait_prior<B_priors>(B_shared_pipeline);

        // print_matrix<BLOCK_TILE_SIZE_K, BLOCK_TILE_SIZE_X>(shared_B[stage]);

        __syncthreads();

        // #pragma unroll
        for (size_t kk{0}; kk < BLOCK_TILE_SIZE_K; ++kk) {
            // we need to start filling the one matrix cache
#pragma unroll
            for (size_t y_cache_idx{0}; y_cache_idx < NUM_CACHES_PER_WARP_Y; ++y_cache_idx) {
                // Here we calculate the row in the shared block based on the warp coordinates
                // and the thread coordinates

                // To calculate the row we first multiply the warp block y coordinate by the
                // Warp y dimension scale on the grid scale this is equivalent to doing blockIdx.y * blockDim.y
                // Next based on what cache we are in we need to skip that many rows. We do this by multiplying the
                // y_cache_idx by (WARP_TILE_SIZE_Y / NUM_CACHES_PER_WARP_Y) this value is equivalent too
                // (THREAD_TILE_SIZE_Y * NUM_THREADS_PER_WARP_Y) which is how many distinct rows are processed per
                // one cache fill of warp tile. Finally, we add the row that this thread is part of in the warp.

                // each thread loads TILE_SIZE_Y
                // assuming this configuration NUM_THREADS_PER_WARP_X = 4, and NUM_THREADS_PER_WARP_Y = 8
                // we can assume this load pattern
                // Threads [0 to 3] load rows [0 to 7], Threads [4 to 7] load rows [8 to 15] ...
                // Threads [28 to 31] load rows [54 to 63], this would result in a bank conflict for each
                // new warp_row and a broadcast for all threads in warp row, but luckily
                // the shared memory is transposed resulting in only broadcasts
                const size_t one_shared_row_idx{
                    warp_row_idx * WARP_TILE_SIZE_Y +
                    y_cache_idx * (WARP_TILE_SIZE_Y / NUM_CACHES_PER_WARP_Y) +
                    thread_idx_in_warp_row * THREAD_TILE_SIZE_Y
                };

                const auto one_shared_ptr{
                    reinterpret_cast<int4 *>(&shared_A_T[stage][kk][one_shared_row_idx])
                };

                auto tile_ptr{
                    reinterpret_cast<int4 *>(&one_cache[y_cache_idx])
                };

                // load into register cache one[y_cache_idx] with vectorized loads
                // #pragma unroll
                for (size_t vy_iter{0}; vy_iter < vectorized_thread_tile_size_y; ++vy_iter)
                    tile_ptr[vy_iter] = one_shared_ptr[vy_iter];
            }

#pragma unroll
            for (size_t x_cache_id{0}; x_cache_id < NUM_CACHES_PER_WARP_X; ++x_cache_id) {
                const size_t two_shared_col_idx{
                    warp_col_idx * WARP_TILE_SIZE_X +
                    x_cache_id * (WARP_TILE_SIZE_X / NUM_CACHES_PER_WARP_X) +
                    thread_idx_in_warp_column * THREAD_TILE_SIZE_X
                };

                const auto two_shared_ptr{
                    reinterpret_cast<int4 *>(&shared_B[stage][kk][two_shared_col_idx])
                };

                auto tile_ptr{
                    reinterpret_cast<int4 *>(&two_cache[x_cache_id])
                };

                // #pragma unroll
                for (size_t vx_iter{0}; vx_iter < vectorized_thread_tile_size_x; ++vx_iter)
                    tile_ptr[vx_iter] = two_shared_ptr[vx_iter];
            }

            // compute intermediates
#pragma unroll
            for (size_t y_cache_idx{0}; y_cache_idx < NUM_CACHES_PER_WARP_Y; ++y_cache_idx) {
                // #pragma unroll
                for (size_t x_cache_idx{0}; x_cache_idx < NUM_CACHES_PER_WARP_X; ++x_cache_idx) {
#pragma unroll
                    for (size_t one_cache_idx{0}; one_cache_idx < THREAD_TILE_SIZE_Y; ++one_cache_idx) {
                        T one_cache_value{one_cache[y_cache_idx][one_cache_idx]};
                        // #pragma unroll
                        for (size_t two_cache_index{0}; two_cache_index < THREAD_TILE_SIZE_X; ++two_cache_index) {
                            intermediates[y_cache_idx][x_cache_idx][one_cache_idx][two_cache_index] +=
                                    one_cache_value * two_cache[x_cache_idx][two_cache_index];
                        }
                    }
                }
            }
        }
        __syncthreads();

#pragma unroll
        for (size_t rel{0}; rel < A_priors; ++rel) {
            A_shared_pipeline.consumer_release();
        }

#pragma unroll
        for (size_t rel{0}; rel < B_priors; ++rel) {
            B_shared_pipeline.consumer_release();
        }

        if (iter < total_iters - 2) {
            load_data_to_shared_async<
                T, BLOCK_TILE_SIZE_X,
                BLOCK_TILE_SIZE_Y, BLOCK_TILE_SIZE_K,
                THREADS_PER_BLOCK, A_priors, B_priors
            >(
                shared_A_T[stage],
                shared_B[stage],
                matrix_A,
                matrix_B,
                k,
                m,
                n,
                leading_dim_A,
                leading_dim_B,
                iter + 2,
                thread_linear_idx,
                A_shared_pipeline,
                B_shared_pipeline
            );
        }

        stage = (stage + 1) % STAGES;
    }

    // vectorized store back into the dest matrix
#pragma unroll
    for (size_t y_cache_idx{0}; y_cache_idx < NUM_CACHES_PER_WARP_Y; ++y_cache_idx) {
#pragma unroll
        for (size_t x_cache_idx{0}; x_cache_idx < NUM_CACHES_PER_WARP_X; ++x_cache_idx) {
            // #pragma unroll
            for (size_t one_cache_idx{0}; one_cache_idx < THREAD_TILE_SIZE_Y; ++one_cache_idx) {
                const size_t dest_row{
                    BLOCK_TILE_SIZE_Y * blockIdx.y +
                    warp_row_idx * WARP_TILE_SIZE_Y +
                    y_cache_idx * (WARP_TILE_SIZE_Y / NUM_CACHES_PER_WARP_Y) +
                    thread_idx_in_warp_row * THREAD_TILE_SIZE_Y + one_cache_idx
                };

                const size_t dest_column{
                    BLOCK_TILE_SIZE_X * blockIdx.x +
                    warp_col_idx * WARP_TILE_SIZE_X +
                    x_cache_idx * (WARP_TILE_SIZE_X / NUM_CACHES_PER_WARP_X) +
                    thread_idx_in_warp_column * THREAD_TILE_SIZE_X
                };

                auto dest_ptr{&matrix_C[dest_row * leading_dim_C + dest_column]};
                T *tile_ptr{&intermediates[y_cache_idx][x_cache_idx][one_cache_idx][0]};

                // #pragma unroll
                for (size_t two_cache_vec_idx{0}; two_cache_vec_idx < vectorized_thread_tile_size_x; ++
                     two_cache_vec_idx) {
                    if (dest_row < m && (
                            dest_column + two_cache_vec_idx * units_per_vector < n)) {
                        // #pragma unroll
                        for (size_t tile_idx{0}; tile_idx < units_per_vector; ++tile_idx) {
                            tile_ptr[tile_idx] = tile_ptr[tile_idx] * alpha + dest_ptr[tile_idx] * beta;
                        }

                        reinterpret_cast<int4 *>(dest_ptr)[two_cache_vec_idx] =
                                reinterpret_cast<int4 *>(tile_ptr)[two_cache_vec_idx];
                    }
                }
            }
        }
    }
}

__device__ __host__ __forceinline__ constexpr bool is_power_of_two(const size_t x) {
    /**
     * first checks if the number is not 0 we do this premptive check since 0 is not a power of 2
     * yet would pass the next check
     *
     * (x & (x - 1)) == 0 this checks if the number is a power of 2
     *
     * all powers of 2 have exactly one 1 bit e.g.:
     * 1 0 0 0 = 1
     * 0 1 0 0 = 2
     * 0 0 1 0 = 4
     * 0 0 0 1 = 8
     *
     * the number before of 2 has to have all 1 bits excpet at the last position
     *
     * 0 0 0 0 = 0
     * 1 0 0 0 = 1
     * 1 1 0 0 = 3
     * 1 1 1 0 = 7
     *
     * so we know a number is a multiple of 2 if the bit wise and of the previous current number == 0
     */
    return x != 0 && (x & (x - 1)) == 0;
}

__device__ __host__ __forceinline__ constexpr size_t next_power_of_two(size_t x) {
    /**
     * We know that 1 - (a power of 2) is all 1 bits so to find the next power of 2
     * our goal is to fill all bits before the highest set bit to 1 and then add
     * 1 to it giving us our next power of 2.
     *
     * We also start off by subtracting 1 to handle the case where the number
     * is already a power of 2
     **/


    if (x == 0) return 1;
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    x |= x >> 32;
    return x + 1;
}

template<
    size_t TILE_SIZE_X,
    size_t TILE_SIZE_Y,
    size_t TILE_SIZE_K,
    size_t PRODUCER_THREADS_PER_BLOCK
>
__device__ __forceinline__ void load_to_shared(
    const size_t iter,
    float A_buffer[TILE_SIZE_Y][TILE_SIZE_K],
    float B_buffer[TILE_SIZE_K][TILE_SIZE_X],
    float *mat_A,
    float *mat_B,
    const size_t M,
    const size_t N,
    const size_t K,
    const size_t thread_linear_idx,
    const size_t A_stride,
    const size_t B_stride) {
    constexpr size_t A_iterations{TILE_SIZE_Y * TILE_SIZE_K / PRODUCER_THREADS_PER_BLOCK};

    for (size_t A_iter{0}; A_iter < A_iterations; ++A_iter) {
        size_t A_buffer_row{(thread_linear_idx + A_iter * PRODUCER_THREADS_PER_BLOCK) / TILE_SIZE_K};
        size_t A_buffer_column{(thread_linear_idx + A_iter * PRODUCER_THREADS_PER_BLOCK) % TILE_SIZE_K};

        float value{0};

        const size_t A_row{blockIdx.y * TILE_SIZE_Y + A_buffer_row};
        const size_t A_column{A_buffer_column + iter * TILE_SIZE_K};

        if (A_row < M && A_column < K)
            value = mat_A[A_row * A_stride + A_column];

        A_buffer[A_buffer_row][A_buffer_column] = value;
    }

    constexpr size_t B_iterations{TILE_SIZE_K * TILE_SIZE_X / PRODUCER_THREADS_PER_BLOCK};

    for (size_t B_iter{0}; B_iter < B_iterations; ++B_iter) {
        size_t B_buffer_row{(thread_linear_idx + B_iter * PRODUCER_THREADS_PER_BLOCK) / TILE_SIZE_X};
        size_t B_buffer_column{(thread_linear_idx + B_iter * PRODUCER_THREADS_PER_BLOCK) % TILE_SIZE_X};

        float value{0};

        const size_t B_row{B_buffer_row + iter * TILE_SIZE_K};
        const size_t B_column{blockIdx.x * TILE_SIZE_X + B_buffer_column};

        if (B_row < K && B_column < N)
            value = mat_B[B_row * B_stride + B_column];

        B_buffer[B_buffer_row][B_buffer_column] = value;
    }
}


template<
    size_t TILE_SIZE_X,
    size_t TILE_SIZE_Y,
    size_t TILE_SIZE_K,
    size_t A_BUFFER_SIZE,
    size_t B_BUFFER_SIZE,
    size_t THREADS_PER_BLOCK
>
__device__ __forceinline__ void load_to_shared_double_buffer(
    const size_t iter,
    float A_buffer[A_BUFFER_SIZE],
    float B_buffer[B_BUFFER_SIZE],
    const float *mat_A,
    const float *mat_B,
    const size_t M,
    const size_t N,
    const size_t K,
    const size_t thread_linear_idx,
    const size_t A_stride,
    const size_t B_stride) {
    constexpr size_t A_iterations{TILE_SIZE_Y * TILE_SIZE_K / THREADS_PER_BLOCK};

    for (size_t A_iter{0}; A_iter < A_iterations; ++A_iter) {
        size_t A_buffer_row{(thread_linear_idx + A_iter * THREADS_PER_BLOCK) / TILE_SIZE_K};
        size_t A_buffer_column{(thread_linear_idx + A_iter * THREADS_PER_BLOCK) % TILE_SIZE_K};

        float value{0};

        const size_t A_row{blockIdx.y * TILE_SIZE_Y + A_buffer_row};
        const size_t A_column{A_buffer_column + iter * TILE_SIZE_K};

        if (A_row < M && A_column < K)
            value = mat_A[A_row * A_stride + A_column];

        A_buffer[A_buffer_row * TILE_SIZE_K + A_buffer_column] = value;
    }

    constexpr size_t B_iterations{TILE_SIZE_K * TILE_SIZE_X / THREADS_PER_BLOCK};

    for (size_t B_iter{0}; B_iter < B_iterations; ++B_iter) {
        size_t B_buffer_row{(thread_linear_idx + B_iter * THREADS_PER_BLOCK) / TILE_SIZE_X};
        size_t B_buffer_column{(thread_linear_idx + B_iter * THREADS_PER_BLOCK) % TILE_SIZE_X};

        float value{0};

        const size_t B_row{B_buffer_row + iter * TILE_SIZE_K};
        const size_t B_column{blockIdx.x * TILE_SIZE_X + B_buffer_column};

        if (B_row < K && B_column < N)
            value = mat_B[B_row * B_stride + B_column];

        B_buffer[B_buffer_row * TILE_SIZE_X + B_buffer_column] = value;
    }
}

template<size_t TILE_SIZE_X, size_t TILE_SIZE_Y, size_t TILE_SIZE_K>
__device__ __forceinline__ void consumer(
    const size_t iterations,
    barrier *ready,
    barrier *filled,
    float A_buffer[2][TILE_SIZE_Y][TILE_SIZE_K],
    float B_buffer[2][TILE_SIZE_K][TILE_SIZE_X],
    float *mat_C,
    const size_t thread_linear_idx,
    const size_t C_stride,
    const size_t M,
    const size_t N) {
    // signal we are ready for the initial shared memory filling
    barrier::arrival_token token1{ready[0].arrive()};
    barrier::arrival_token token2{ready[1].arrive()};

    float partial{0};

    size_t row{thread_linear_idx / TILE_SIZE_X};
    size_t column{thread_linear_idx % TILE_SIZE_X};

    const size_t C_row{TILE_SIZE_Y * blockIdx.y + row};
    const size_t C_column{blockIdx.x * TILE_SIZE_X + column};

    for (size_t iter{0}; iter < iterations; ++iter) {
        // alternate the buffers being used
        const size_t selected_buffer{iter % 2};

        // wait for that buffer to be ready
        filled[selected_buffer].arrive_and_wait();

        // consumption

        for (size_t k{0}; k < TILE_SIZE_K; ++k)
            partial += A_buffer[selected_buffer][row][k] * B_buffer[selected_buffer][k][column];

        // buffer is ready to be filled again
        barrier::arrival_token token{ready[selected_buffer].arrive()};
    }

    if (C_row < M && C_column < N) {
        mat_C[C_row * C_stride + C_column] = partial;
    }
}

template<
    size_t TILE_SIZE_X,
    size_t TILE_SIZE_Y,
    size_t TILE_SIZE_K,
    size_t PRODUCER_THREADS_PER_BLOCK
>
__device__ __forceinline__ void producer(
    const size_t iterations,
    barrier *ready,
    barrier *filled,
    float A_buffer[2][TILE_SIZE_Y][TILE_SIZE_K],
    float B_buffer[2][TILE_SIZE_K][TILE_SIZE_X],
    float *mat_A,
    float *mat_B,
    const size_t M,
    const size_t N,
    const size_t K,
    const size_t thread_linear_idx,
    const size_t A_stride,
    const size_t B_stride) {
    for (size_t iter{0}; iter < iterations; ++iter) {
        const size_t selected_buffer{iter % 2};
        ready[selected_buffer].arrive_and_wait();

        // fill shared memory
        load_to_shared<TILE_SIZE_X, TILE_SIZE_Y, TILE_SIZE_K, PRODUCER_THREADS_PER_BLOCK>(
            iter,
            A_buffer[selected_buffer],
            B_buffer[selected_buffer],
            mat_A,
            mat_B,
            M,
            N,
            K,
            thread_linear_idx,
            A_stride,
            B_stride
        );

        // wait for that buffer to be ready
        barrier::arrival_token token{filled[selected_buffer].arrive()};
    }
}

template<size_t TILE_SIZE_X, size_t TILE_SIZE_K>
__device__ __forceinline__ void accumulate(
    float &partial,
    const uintptr_t A_block_load_addr,
    const uintptr_t B_block_load_addr) {
    const auto A_load_ptr{reinterpret_cast<float *>(A_block_load_addr)};
    const auto B_load_ptr{reinterpret_cast<float *>(B_block_load_addr)};

    for (size_t k{0}; k < TILE_SIZE_K; ++k)
        partial += A_load_ptr[threadIdx.y * TILE_SIZE_K + k] * B_load_ptr[k * TILE_SIZE_X + threadIdx.x];
}

template<
    size_t THREADS_PER_BLOCK,
    size_t TILE_SIZE_X,
    size_t TILE_SIZE_Y,
    size_t TILE_SIZE_K>
__global__ void gemm_double_buffering(
    float *mat_A,
    float *mat_B,
    float *mat_C,
    const size_t M,
    const size_t N,
    const size_t K,
    const size_t A_stride,
    const size_t B_stride,
    const size_t C_stride) {
    // implementation derived from https://salykova.github.io/sgemm-gpu section 5.1

    constexpr size_t A_buffer_size{next_power_of_two(2 * TILE_SIZE_Y * TILE_SIZE_K * sizeof(float))};
    constexpr size_t B_buffer_size{next_power_of_two(2 * TILE_SIZE_K * TILE_SIZE_X * sizeof(float))};

    __shared__ float __align__(A_buffer_size) A_buffer[A_buffer_size / sizeof(float)];
    __shared__ float __align__(B_buffer_size) B_buffer[B_buffer_size / sizeof(float)];

    const size_t iterations{ceil_div(K, TILE_SIZE_K) - 1};

    const size_t thread_linear_idx{threadIdx.y * TILE_SIZE_X + threadIdx.x};
    const size_t C_row{blockIdx.y * TILE_SIZE_Y + threadIdx.y};
    const size_t C_column{blockIdx.x * TILE_SIZE_X + threadIdx.x};

    load_to_shared_double_buffer<
        TILE_SIZE_X,
        TILE_SIZE_Y,
        TILE_SIZE_K,
        A_buffer_size,
        B_buffer_size,
        THREADS_PER_BLOCK>(
        0,
        A_buffer,
        B_buffer,
        mat_A,
        mat_B,
        M,
        N,
        K,
        thread_linear_idx,
        A_stride,
        B_stride
    );

    __syncthreads();

    auto A_block_store_addr = reinterpret_cast<uintptr_t>(A_buffer);
    auto A_block_load_addr = reinterpret_cast<uintptr_t>(A_buffer);

    auto B_block_store_addr = reinterpret_cast<uintptr_t>(B_buffer);
    auto B_block_load_addr = reinterpret_cast<uintptr_t>(B_buffer);

    constexpr uintptr_t flip_bits_A{A_buffer_size / 2};
    constexpr uintptr_t flip_bits_B{A_buffer_size / 2};

    A_block_store_addr ^= flip_bits_A;
    B_block_store_addr ^= flip_bits_B;

    float partial{0};

    for (size_t iter{0}; iter < iterations; ++iter) {
        auto A_store_ptr{reinterpret_cast<float *>(A_block_store_addr)};
        auto B_store_ptr{reinterpret_cast<float *>(B_block_store_addr)};

        load_to_shared_double_buffer<
            TILE_SIZE_X,
            TILE_SIZE_Y,
            TILE_SIZE_K,
            A_buffer_size,
            B_buffer_size,
            THREADS_PER_BLOCK>(
            iter + 1,
            A_store_ptr,
            B_store_ptr,
            mat_A,
            mat_B,
            M,
            N,
            K,
            thread_linear_idx,
            A_stride,
            B_stride
        );

        accumulate<TILE_SIZE_X, TILE_SIZE_K>(partial, A_block_load_addr, B_block_load_addr);

        A_block_load_addr ^= flip_bits_A;
        A_block_store_addr ^= flip_bits_A;

        B_block_load_addr ^= flip_bits_B;
        B_block_store_addr ^= flip_bits_B;

        __syncthreads();
    }

    accumulate<TILE_SIZE_X, TILE_SIZE_K>(partial, A_block_load_addr, B_block_load_addr);

    if (C_row < M && C_column < N) {
        mat_C[C_row * C_stride + C_column] = partial;
    }
}


template<
    size_t CONSUMER_WARPS,
    size_t PRODUCER_WARPS,
    size_t WARP_SIZE = 32,
    size_t TILE_SIZE_X,
    size_t TILE_SIZE_Y,
    size_t TILE_SIZE_K>
__global__ void producer_consumer_pattern(
    float *mat_A,
    float *mat_B,
    float *mat_C,
    const size_t M,
    const size_t N,
    const size_t K,
    const size_t A_stride,
    const size_t B_stride,
    const size_t C_stride) {
    // CONSUMER_WARPS will handle processing the data
    // PRODUCER_WARPS will handle writing the data to memory

    // CONSUMER WARP signal if the array is ready to be filled
    // PRODUCER WARPS signal if the array is ready to be consumed

    // double buffer
    __shared__ float A_buffer[2][TILE_SIZE_Y][TILE_SIZE_K];
    __shared__ float B_buffer[2][TILE_SIZE_K][TILE_SIZE_X];
    __shared__ barrier bar[4];

    auto block = cooperative_groups::this_thread_block();
    constexpr size_t producer_threads_per_block{WARP_SIZE * PRODUCER_WARPS};
    constexpr size_t consumer_threads_per_block{WARP_SIZE * CONSUMER_WARPS};

    const size_t total_iters{ceil_div(K, TILE_SIZE_K)};

    // initialization
    if (block.thread_rank() == 0) {
        // tracks if a buffer is ready to be filled

        // printf("%d, %lu \n", block.size(), producer_threads_per_block + consumer_threads_per_block);
        init(&bar[0], producer_threads_per_block + consumer_threads_per_block);
        init(&bar[1], producer_threads_per_block + consumer_threads_per_block);

        // tracks if a buffer is ready to be consumed
        init(&bar[2], producer_threads_per_block + consumer_threads_per_block);
        init(&bar[3], producer_threads_per_block + consumer_threads_per_block);
    }

    block.sync();

    if (block.thread_rank() < producer_threads_per_block) {
        producer<TILE_SIZE_X,
            TILE_SIZE_Y, TILE_SIZE_K,
            producer_threads_per_block>(
            total_iters,
            bar,
            bar + 2,
            A_buffer,
            B_buffer,
            mat_A,
            mat_B,
            M,
            N,
            K,
            block.thread_rank(),
            A_stride,
            B_stride);
    } else {
        const size_t thread_linear_idx{block.thread_rank() - producer_threads_per_block};
        consumer<TILE_SIZE_X,
            TILE_SIZE_Y, TILE_SIZE_K>(
            total_iters,
            bar,
            bar + 2,
            A_buffer,
            B_buffer,
            mat_C,
            thread_linear_idx,
            C_stride,
            M,
            N
        );
    }
}

template<
    size_t TILE_SIZE_X,
    size_t TILE_SIZE_Y,
    size_t TILE_SIZE_K,
    size_t STAGES = 2>
__global__ void gemm_2DBT_async(
    float *mat_A,
    float *mat_B,
    float *mat_C,
    const size_t M,
    const size_t N,
    const size_t K,
    const size_t A_stride,
    const size_t B_stride,
    const size_t C_stride) {
    __shared__ float A_buffer[STAGES][TILE_SIZE_Y][TILE_SIZE_K];
    __shared__ float B_buffer[STAGES][TILE_SIZE_K][TILE_SIZE_X];

    const size_t total_iters{ceil_div(K, TILE_SIZE_K)};
    const size_t thread_linear_idx{TILE_SIZE_X * threadIdx.y + threadIdx.x};

    constexpr size_t total_threads{TILE_SIZE_X * TILE_SIZE_Y};

    constexpr size_t A_priors{TILE_SIZE_Y * TILE_SIZE_K / total_threads};
    constexpr size_t B_priors{TILE_SIZE_X * TILE_SIZE_K / total_threads};

    const size_t C_col{TILE_SIZE_X * blockIdx.x + threadIdx.x};
    const size_t C_row{TILE_SIZE_Y * blockIdx.y + threadIdx.y};

    cuda::pipeline<cuda::thread_scope_thread> A_shared_pipeline{cuda::make_pipeline()};
    cuda::pipeline<cuda::thread_scope_thread> B_shared_pipeline{cuda::make_pipeline()};

    for (size_t stage{0}; stage < STAGES; ++stage) {
        load_data_to_shared_async<
            float,
            TILE_SIZE_X,
            TILE_SIZE_Y,
            TILE_SIZE_K,
            total_threads,
            A_priors,
            B_priors,
            0,
            0,
            0,
            false
        >(
            A_buffer[stage],
            B_buffer[stage],
            mat_A,
            mat_B,
            K,
            M,
            N,
            A_stride,
            B_stride,
            stage,
            thread_linear_idx,
            A_shared_pipeline,
            B_shared_pipeline
        );
    }

    size_t stage{0};
    float partial{0.f};

    for (size_t iter{0}; iter < total_iters; ++iter) {
        cuda::pipeline_consumer_wait_prior<A_priors>(A_shared_pipeline);
        cuda::pipeline_consumer_wait_prior<B_priors>(B_shared_pipeline);

        __syncthreads();

        for (size_t k{0}; k < TILE_SIZE_K; ++k) {
            partial += A_buffer[stage][threadIdx.y][k] * B_buffer[stage][k][threadIdx.x];
        }
        __syncthreads();

#pragma unroll
        for (size_t rel{0}; rel < A_priors; ++rel) {
            A_shared_pipeline.consumer_release();
        }

#pragma unroll
        for (size_t rel{0}; rel < B_priors; ++rel) {
            B_shared_pipeline.consumer_release();
        }

        load_data_to_shared_async<
            float,
            TILE_SIZE_X,
            TILE_SIZE_Y,
            TILE_SIZE_K,
            total_threads,
            A_priors,
            B_priors,
            0,
            0,
            0,
            false
        >(
            A_buffer[stage],
            B_buffer[stage],
            mat_A,
            mat_B,
            K,
            M,
            N,
            A_stride,
            B_stride,
            iter + 2,
            thread_linear_idx,
            A_shared_pipeline,
            B_shared_pipeline
        );

        stage = (stage + 1) % STAGES;
    }

    if (C_col < N && C_row < M)
        mat_C[C_stride * C_row + C_col] = partial;
}


// testing section
//---------------------------------------------------------------------------------------------------------------------

template<
    size_t TILE_SIZE_X,
    size_t TILE_SIZE_Y,
    size_t TILE_SIZE_K>
__global__ void gemm_2DBT(
    float *mat_A,
    float *mat_B,
    float *mat_C,
    const size_t M,
    const size_t N,
    const size_t K,
    const size_t A_stride,
    const size_t B_stride,
    const size_t C_stride) {
    __shared__ float A_buffer[TILE_SIZE_Y][TILE_SIZE_K];
    __shared__ float B_buffer[TILE_SIZE_K][TILE_SIZE_X];

    const size_t total_iters{ceil_div(K, TILE_SIZE_K)};
    const size_t thread_linear_idx{TILE_SIZE_X * threadIdx.y + threadIdx.x};

    constexpr size_t total_threads{TILE_SIZE_X * TILE_SIZE_Y};

    const size_t C_col{TILE_SIZE_X * blockIdx.x + threadIdx.x};
    const size_t C_row{TILE_SIZE_Y * blockIdx.y + threadIdx.y};

    float partial{0.f};

    for (size_t iter{0}; iter < total_iters; ++iter) {
        load_to_shared<TILE_SIZE_X, TILE_SIZE_Y, TILE_SIZE_K, total_threads>(
            iter,
            A_buffer,
            B_buffer,
            mat_A,
            mat_B,
            M,
            N,
            K,
            thread_linear_idx,
            A_stride,
            B_stride);

        __syncthreads();

        for (size_t k{0}; k < TILE_SIZE_K; ++k) {
            partial += A_buffer[threadIdx.y][k] * B_buffer[k][threadIdx.x];
        }
        __syncthreads();
    }

    if (C_col < N && C_row < M)
        mat_C[C_stride * C_row + C_col] = partial;
}

void cpu_matmul_naive(
    const float *mat_A,
    const float *mat_B,
    float *mat_C,
    const size_t M,
    const size_t N,
    const size_t K,
    const size_t A_stride,
    const size_t B_stride,
    const size_t C_stride) {
    for (size_t i{0}; i < M; ++i) {
        for (size_t j{0}; j < N; ++j) {
            for (size_t k{0}; k < K; ++k) {
                mat_C[i * C_stride + j] +=
                        mat_A[A_stride * i + k] * mat_B[B_stride * k + j];
            }
        }
    }
}

void test_equivalency(
    const float *expected,
    const float *result,
    const size_t M,
    const size_t N,
    const size_t stride) {
    for (size_t i{0}; i < M; ++i) {
        for (size_t j{0}; j < N; ++j) {
            assert(expected[i * stride + j] == result[i * stride + j]);
        }
    }
}

float generate_random_float(const float min_val, const float max_val) {
    static std::random_device rd;
    static std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(min_val, max_val);
    return dis(gen);
}

int generate_random_int(const int min_val, const int max_val) {
    static std::random_device rd;
    static std::mt19937 gen(rd());
    std::uniform_int_distribution dis(min_val, max_val);
    return dis(gen);
}

void fill_matrix(
    float *matrix,
    const size_t M,
    const size_t N,
    const size_t stride,
    const float min_val,
    const float max_val) {
    for (size_t i{0}; i < M; ++i) {
        for (size_t j{0}; j < N; ++j) {
            matrix[i * stride + j] = generate_random_float(min_val, max_val);
        }
    }
}

void fill_matrix_w(
    float *matrix,
    const size_t M,
    const size_t N,
    const size_t stride,
    const int min_val,
    const int max_val) {
    for (size_t i{0}; i < M; ++i) {
        for (size_t j{0}; j < N; ++j) {
            matrix[i * stride + j] = static_cast<float>(generate_random_int(min_val, max_val));
        }
    }
}


void print_matrix(
    const float *matrix,
    const size_t M,
    const size_t N,
    const size_t stride) {
    for (size_t i{0}; i < M; ++i) {
        std::cout << "[";
        for (size_t j{0}; j < N; ++j) {
            std::cout << matrix[i * stride + j] << " ";
        }
        std::cout << "]\n";
    }
}

void test_double_buffer_gemm() {
    // const auto data{new float[10 * 8]};
    //
    // print_matrix(data, 10, 8, 8);

    auto host_A{new float[211 * 35]};
    auto host_B{new float[35 * 68]};
    auto host_C{new float[211 * 68]};
    auto host_C_2{new float[211 * 68]};

    float *dev_A;
    float *dev_B;
    float *dev_C;

    fill_matrix_w(host_A, 211, 35, 35, -100, 100);
    fill_matrix_w(host_B, 35, 68, 68, -100, 100);
    fill_matrix_w(host_C, 211, 68, 68, 0, 0);

    hipMalloc(&dev_A, 211 * 35 * sizeof(float));
    hipMalloc(&dev_B, 35 * 68 * sizeof(float));
    hipMalloc(&dev_C, 211 * 68 * sizeof(float));

    hipMemcpy(dev_A, host_A, 211 * 35 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, host_B, 35 * 68 * sizeof(float), hipMemcpyHostToDevice);

    cpu_matmul_naive(
        host_A,
        host_B,
        host_C,
        211,
        68,
        35,
        35,
        68,
        68
    );

    constexpr size_t TILE_SIZE_X{16};
    constexpr size_t TILE_SIZE_Y{16};
    constexpr size_t TILE_SIZE_K{16};

    constexpr size_t THREADS_PER_BLOCK{TILE_SIZE_X * TILE_SIZE_Y};

    static_assert(TILE_SIZE_K * TILE_SIZE_X % THREADS_PER_BLOCK == 0);
    static_assert(TILE_SIZE_K * TILE_SIZE_Y % THREADS_PER_BLOCK == 0);

    constexpr dim3 block_dim(TILE_SIZE_X, TILE_SIZE_Y);
    const dim3 grid_dim(ceil_div(68, TILE_SIZE_X), ceil_div(211, TILE_SIZE_Y));

    gemm_double_buffering<
        THREADS_PER_BLOCK,
        TILE_SIZE_X,
        TILE_SIZE_Y,
        TILE_SIZE_K><<<grid_dim, block_dim>>>(
        dev_A,
        dev_B,
        dev_C,
        211,
        68,
        35,
        35,
        68,
        68
    );

    hipDeviceSynchronize();

    // Check for errors in kernel execution
    if (const hipError_t error = hipGetLastError(); error != hipSuccess)
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;

    hipMemcpy(host_C_2, dev_C, 211 * 68 * sizeof(float), hipMemcpyDeviceToHost);

    test_equivalency(host_C, host_C_2, 211, 68, 68);

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    delete []host_A;
    delete []host_B;
    delete []host_C;
    delete []host_C_2;
}

void test_gemm_2DBT() {
    // const auto data{new float[10 * 8]};
    //
    // print_matrix(data, 10, 8, 8);

    auto host_A{new float[211 * 35]};
    auto host_B{new float[35 * 68]};
    auto host_C{new float[211 * 68]};
    auto host_C_2{new float[211 * 68]};

    float *dev_A;
    float *dev_B;
    float *dev_C;

    fill_matrix_w(host_A, 211, 35, 35, -100, 100);
    fill_matrix_w(host_B, 35, 68, 68, -100, 100);
    fill_matrix_w(host_C, 211, 68, 68, 0, 0);

    hipMalloc(&dev_A, 211 * 35 * sizeof(float));
    hipMalloc(&dev_B, 35 * 68 * sizeof(float));
    hipMalloc(&dev_C, 211 * 68 * sizeof(float));

    hipMemcpy(dev_A, host_A, 211 * 35 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, host_B, 35 * 68 * sizeof(float), hipMemcpyHostToDevice);

    cpu_matmul_naive(
        host_A,
        host_B,
        host_C,
        211,
        68,
        35,
        35,
        68,
        68
    );

    constexpr size_t TILE_SIZE_X{16};
    constexpr size_t TILE_SIZE_Y{16};
    constexpr size_t TILE_SIZE_K{16};

    constexpr size_t THREADS_PER_BLOCK{TILE_SIZE_X * TILE_SIZE_Y};

    static_assert(TILE_SIZE_K * TILE_SIZE_X % THREADS_PER_BLOCK == 0);
    static_assert(TILE_SIZE_K * TILE_SIZE_Y % THREADS_PER_BLOCK == 0);

    constexpr dim3 block_dim(TILE_SIZE_X, TILE_SIZE_Y);
    const dim3 grid_dim(ceil_div(68, TILE_SIZE_X), ceil_div(211, TILE_SIZE_Y));

    gemm_2DBT<
        TILE_SIZE_X,
        TILE_SIZE_Y,
        TILE_SIZE_K><<<grid_dim, block_dim>>>(
        dev_A,
        dev_B,
        dev_C,
        211,
        68,
        35,
        35,
        68,
        68
    );

    hipDeviceSynchronize();

    // Check for errors in kernel execution
    if (const hipError_t error = hipGetLastError(); error != hipSuccess)
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;

    hipMemcpy(host_C_2, dev_C, 211 * 68 * sizeof(float), hipMemcpyDeviceToHost);

    test_equivalency(host_C, host_C_2, 211, 68, 68);

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    delete []host_A;
    delete []host_B;
    delete []host_C;
    delete []host_C_2;
}

void test_gemm_2DBT_async() {
    // const auto data{new float[10 * 8]};
    //
    // print_matrix(data, 10, 8, 8);

    auto host_A{new float[211 * 35]};
    auto host_B{new float[35 * 68]};
    auto host_C{new float[211 * 68]};
    auto host_C_2{new float[211 * 68]};

    float *dev_A;
    float *dev_B;
    float *dev_C;

    fill_matrix_w(host_A, 211, 35, 35, -100, 100);
    fill_matrix_w(host_B, 35, 68, 68, -100, 100);
    fill_matrix_w(host_C, 211, 68, 68, 0, 0);

    hipMalloc(&dev_A, 211 * 35 * sizeof(float));
    hipMalloc(&dev_B, 35 * 68 * sizeof(float));
    hipMalloc(&dev_C, 211 * 68 * sizeof(float));

    hipMemcpy(dev_A, host_A, 211 * 35 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, host_B, 35 * 68 * sizeof(float), hipMemcpyHostToDevice);

    cpu_matmul_naive(
        host_A,
        host_B,
        host_C,
        211,
        68,
        35,
        35,
        68,
        68
    );

    constexpr size_t TILE_SIZE_X{16};
    constexpr size_t TILE_SIZE_Y{16};
    constexpr size_t TILE_SIZE_K{16};

    constexpr size_t THREADS_PER_BLOCK{TILE_SIZE_X * TILE_SIZE_Y};

    static_assert(TILE_SIZE_K * TILE_SIZE_X % THREADS_PER_BLOCK == 0);
    static_assert(TILE_SIZE_K * TILE_SIZE_Y % THREADS_PER_BLOCK == 0);

    constexpr dim3 block_dim(TILE_SIZE_X, TILE_SIZE_Y);
    const dim3 grid_dim(ceil_div(68, TILE_SIZE_X), ceil_div(211, TILE_SIZE_Y));

    gemm_2DBT_async<
        TILE_SIZE_X,
        TILE_SIZE_Y,
        TILE_SIZE_K><<<grid_dim, block_dim>>>(
        dev_A,
        dev_B,
        dev_C,
        211,
        68,
        35,
        35,
        68,
        68
    );

    hipDeviceSynchronize();

    // Check for errors in kernel execution
    if (const hipError_t error = hipGetLastError(); error != hipSuccess)
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;

    hipMemcpy(host_C_2, dev_C, 211 * 68 * sizeof(float), hipMemcpyDeviceToHost);

    test_equivalency(host_C, host_C_2, 211, 68, 68);

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    delete []host_A;
    delete []host_B;
    delete []host_C;
    delete []host_C_2;
}

void test_gemm_2DBT_2DWT_2DTT_vload() {
    auto host_A{new float[211 * 32]};
    auto host_B{new float[32 * 64]};
    auto host_C{new float[211 * 64]};
    auto host_C_2{new float[211 * 64]};

    float *dev_A;
    float *dev_B;
    float *dev_C;

    fill_matrix_w(host_A, 211, 32, 32, -100, 100);
    fill_matrix_w(host_B, 32, 64, 64, -100, 100);
    fill_matrix_w(host_C, 211, 64, 64, 0, 0);

    hipMalloc(&dev_A, 211 * 32 * sizeof(float));
    hipMalloc(&dev_B, 32 * 64 * sizeof(float));
    hipMalloc(&dev_C, 211 * 64 * sizeof(float));

    hipMemcpy(dev_A, host_A, 211 * 32 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, host_B, 32 * 64 * sizeof(float), hipMemcpyHostToDevice);

    cpu_matmul_naive(
        host_A,
        host_B,
        host_C,
        211,
        64,
        32,
        32,
        64,
        64
    );

    constexpr uint BLOCK_TILE_SIZE_X{128};
    constexpr uint BLOCK_TILE_SIZE_Y{128};
    constexpr uint BLOCK_TILE_SIZE_K{16};

    constexpr unsigned int WARP_TILE_SIZE_X{64};
    constexpr unsigned int WARP_TILE_SIZE_Y{64};

    constexpr size_t NUM_WARPS_PER_BLOCK_X{BLOCK_TILE_SIZE_X / WARP_TILE_SIZE_X};
    constexpr size_t NUM_WARPS_PER_BLOCK_Y{BLOCK_TILE_SIZE_Y / WARP_TILE_SIZE_Y};

    static_assert(BLOCK_TILE_SIZE_X % WARP_TILE_SIZE_X == 0);
    static_assert(BLOCK_TILE_SIZE_Y % WARP_TILE_SIZE_Y == 0);

    // The size of the internal register caches
    constexpr uint THREAD_TILE_SIZE_Y{8};
    constexpr uint THREAD_TILE_SIZE_X{8};

    constexpr unsigned int NUM_THREADS_PER_WARP_X{4};
    constexpr unsigned int NUM_THREADS_PER_WARP_Y{8};

    static_assert(NUM_THREADS_PER_WARP_X * NUM_THREADS_PER_WARP_Y == 32);

    // ensure each thread stores the same amount of data in their tiles
    static_assert(WARP_TILE_SIZE_X % (THREAD_TILE_SIZE_X * NUM_THREADS_PER_WARP_X) == 0);
    static_assert(WARP_TILE_SIZE_Y % (THREAD_TILE_SIZE_Y * NUM_THREADS_PER_WARP_Y) == 0);

    const dim3 grid_dim{
        ceil_div(64, BLOCK_TILE_SIZE_X),
        ceil_div(211, BLOCK_TILE_SIZE_Y)
    };

    constexpr size_t NUM_THREADS_PER_BLOCK{32 * NUM_WARPS_PER_BLOCK_X * NUM_WARPS_PER_BLOCK_Y};
    constexpr dim3 block_dim(NUM_THREADS_PER_BLOCK);

    gemm_2DBT_2DWT_2DTT_vload<
        float,
        BLOCK_TILE_SIZE_X,
        BLOCK_TILE_SIZE_Y,
        BLOCK_TILE_SIZE_K,
        WARP_TILE_SIZE_X,
        WARP_TILE_SIZE_Y,
        THREAD_TILE_SIZE_X,
        THREAD_TILE_SIZE_Y,
        NUM_THREADS_PER_WARP_X,
        NUM_THREADS_PER_WARP_Y><<<grid_dim, block_dim>>>(
        dev_A,
        dev_B,
        dev_C,
        1,
        1,
        211,
        64,
        32,
        32,
        64,
        64
    );

    hipDeviceSynchronize();

    // Check for errors in kernel execution
    if (const hipError_t error = hipGetLastError(); error != hipSuccess)
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;

    hipMemcpy(host_C_2, dev_C, 211 * 64 * sizeof(float), hipMemcpyDeviceToHost);

    test_equivalency(host_C, host_C_2, 211, 64, 64);

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    delete []host_A;
    delete []host_B;
    delete []host_C;
    delete []host_C_2;
}

void test_async_load_gemm() {
    // const auto data{new float[10 * 8]};
    //
    // print_matrix(data, 10, 8, 8);

    constexpr size_t m{211};
    constexpr size_t n{68};
    constexpr size_t k{35};

    auto host_A{new float[m * k]};
    auto host_B{new float[k * n]};
    auto host_C{new float[m * n]};
    auto host_C_2{new float[m * n]};

    float *dev_A;
    float *dev_B;
    float *dev_C;

    fill_matrix_w(host_A, m, k, k, -100, 100);
    fill_matrix_w(host_B, k, n, n, -100, 100);
    fill_matrix_w(host_C, m, n, n, 0, 0);

    hipMalloc(&dev_A, m * k * sizeof(float));
    hipMalloc(&dev_B, k * n * sizeof(float));
    hipMalloc(&dev_C, m * n * sizeof(float));

    hipMemcpy(dev_A, host_A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, host_B, k * n * sizeof(float), hipMemcpyHostToDevice);

    cpu_matmul_naive(
        host_A,
        host_B,
        host_C,
        m,
        n,
        k,
        k,
        n,
        n
    );

    constexpr uint BLOCK_TILE_SIZE_X{128};
    constexpr uint BLOCK_TILE_SIZE_Y{128};
    constexpr uint BLOCK_TILE_SIZE_K{16};

    constexpr unsigned int WARP_TILE_SIZE_X{64};
    constexpr unsigned int WARP_TILE_SIZE_Y{64};

    constexpr size_t NUM_WARPS_PER_BLOCK_X{BLOCK_TILE_SIZE_X / WARP_TILE_SIZE_X};
    constexpr size_t NUM_WARPS_PER_BLOCK_Y{BLOCK_TILE_SIZE_Y / WARP_TILE_SIZE_Y};

    static_assert(BLOCK_TILE_SIZE_X % WARP_TILE_SIZE_X == 0);
    static_assert(BLOCK_TILE_SIZE_Y % WARP_TILE_SIZE_Y == 0);

    // The size of the internal register caches
    constexpr uint THREAD_TILE_SIZE_Y{8};
    constexpr uint THREAD_TILE_SIZE_X{8};

    constexpr unsigned int NUM_THREADS_PER_WARP_X{4};
    constexpr unsigned int NUM_THREADS_PER_WARP_Y{8};

    static_assert(NUM_THREADS_PER_WARP_X * NUM_THREADS_PER_WARP_Y == 32);

    // ensure each thread stores the same amount of data in their tiles
    static_assert(WARP_TILE_SIZE_X % (THREAD_TILE_SIZE_X * NUM_THREADS_PER_WARP_X) == 0);
    static_assert(WARP_TILE_SIZE_Y % (THREAD_TILE_SIZE_Y * NUM_THREADS_PER_WARP_Y) == 0);

    const dim3 grid_dim{
        ceil_div(n, BLOCK_TILE_SIZE_X),
        ceil_div(m, BLOCK_TILE_SIZE_Y)
    };

    constexpr size_t NUM_THREADS_PER_BLOCK{32 * NUM_WARPS_PER_BLOCK_X * NUM_WARPS_PER_BLOCK_Y};
    constexpr dim3 block_dim(NUM_THREADS_PER_BLOCK);

    gemm_2DBT_2DWT_2DTT_async_load<
        float,
        BLOCK_TILE_SIZE_X,
        BLOCK_TILE_SIZE_Y,
        BLOCK_TILE_SIZE_K,
        WARP_TILE_SIZE_X,
        WARP_TILE_SIZE_Y,
        THREAD_TILE_SIZE_X,
        THREAD_TILE_SIZE_Y,
        NUM_THREADS_PER_WARP_X,
        NUM_THREADS_PER_WARP_Y><<<grid_dim, block_dim>>>(
        dev_A,
        dev_B,
        dev_C,
        1,
        1,
        m,
        n,
        k,
        k,
        n,
        n
    );

    hipDeviceSynchronize();

    // Check for errors in kernel execution
    if (const hipError_t error = hipGetLastError(); error != hipSuccess)
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;

    hipMemcpy(host_C_2, dev_C, m * n * sizeof(float), hipMemcpyDeviceToHost);

    test_equivalency(host_C, host_C_2, m, n, n);

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    delete []host_A;
    delete []host_B;
    delete []host_C;
    delete []host_C_2;
}

void test_double_buffer() {
    // const auto data{new float[10 * 8]};
    //
    // print_matrix(data, 10, 8, 8);

    auto host_A{new float[211 * 35]};
    auto host_B{new float[35 * 68]};
    auto host_C{new float[211 * 68]};
    auto host_C_2{new float[211 * 68]};

    float *dev_A;
    float *dev_B;
    float *dev_C;

    fill_matrix_w(host_A, 211, 35, 35, -100, 100);
    fill_matrix_w(host_B, 35, 68, 68, -100, 100);
    fill_matrix_w(host_C, 211, 68, 68, 0, 0);

    hipMalloc(&dev_A, 211 * 35 * sizeof(float));
    hipMalloc(&dev_B, 35 * 68 * sizeof(float));
    hipMalloc(&dev_C, 211 * 68 * sizeof(float));

    hipMemcpy(dev_A, host_A, 211 * 35 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, host_B, 35 * 68 * sizeof(float), hipMemcpyHostToDevice);

    cpu_matmul_naive(
        host_A,
        host_B,
        host_C,
        211,
        68,
        35,
        35,
        68,
        68
    );

    constexpr size_t TILE_SIZE_X{16};
    constexpr size_t TILE_SIZE_Y{16};
    constexpr size_t TILE_SIZE_K{16};
    constexpr size_t CONSUMER_WARPS{16 * 16 / 32};
    constexpr size_t PRODUCER_WARPS{4};

    constexpr size_t THREADS_PER_BLOCK{TILE_SIZE_X * TILE_SIZE_Y};

    static_assert(TILE_SIZE_K * TILE_SIZE_X % THREADS_PER_BLOCK == 0);
    static_assert(TILE_SIZE_K * TILE_SIZE_Y % THREADS_PER_BLOCK == 0);

    constexpr dim3 block_dim(PRODUCER_WARPS * 32 + CONSUMER_WARPS * 32);
    const dim3 grid_dim(ceil_div(68, TILE_SIZE_X), ceil_div(211, TILE_SIZE_Y));

    producer_consumer_pattern<
        CONSUMER_WARPS,
        PRODUCER_WARPS,
        32,
        TILE_SIZE_X,
        TILE_SIZE_Y,
        TILE_SIZE_K><<<grid_dim, block_dim>>>(
        dev_A,
        dev_B,
        dev_C,
        211,
        68,
        35,
        35,
        68,
        68
    );

    hipDeviceSynchronize();

    // Check for errors in kernel execution
    if (const hipError_t error = hipGetLastError(); error != hipSuccess)
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;

    hipMemcpy(host_C_2, dev_C, 211 * 68 * sizeof(float), hipMemcpyDeviceToHost);

    test_equivalency(host_C, host_C_2, 211, 68, 68);

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    delete []host_A;
    delete []host_B;
    delete []host_C;
    delete []host_C_2;
}

void flush_l2_cache() {
    int dev_id{};
    int m_l2_size{};
    void *buffer;
    hipGetDevice(&dev_id);
    hipDeviceGetAttribute(&m_l2_size, hipDeviceAttributeL2CacheSize, dev_id);
    if (m_l2_size > 0) {
        hipMalloc(&buffer, static_cast<std::size_t>(m_l2_size));
        int *m_l2_buffer = reinterpret_cast<int *>(buffer);
        hipMemsetAsync(m_l2_buffer, 0, static_cast<std::size_t>(m_l2_size));
        hipFree(m_l2_buffer);
    }

    // Check for errors in kernel execution
    if (const hipError_t error = hipGetLastError(); error != hipSuccess)
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
}

void time_2DBT() {
    auto host_A{new float[4096 * 4096]};
    auto host_B{new float[4096 * 4096]};

    float *dev_A;
    float *dev_B;
    float *dev_C;

    fill_matrix(host_A, 4096, 4096, 4096, -100.f, 100.f);
    fill_matrix(host_B, 4096, 4096, 4096, -100.f, 100.f);

    hipMalloc(&dev_A, 4096 * 4096 * sizeof(float));
    hipMalloc(&dev_B, 4096 * 4096 * sizeof(float));
    hipMalloc(&dev_C, 4096 * 4096 * sizeof(float));

    hipMemcpy(dev_A, host_A, 4096 * 4096 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, host_B, 4096 * 4096 * sizeof(float), hipMemcpyHostToDevice);

    constexpr size_t TILE_SIZE_X{16};
    constexpr size_t TILE_SIZE_Y{16};
    constexpr size_t TILE_SIZE_K{16};

    constexpr size_t THREADS_PER_BLOCK{TILE_SIZE_X * TILE_SIZE_Y};

    static_assert(TILE_SIZE_K * TILE_SIZE_X % THREADS_PER_BLOCK == 0);
    static_assert(TILE_SIZE_K * TILE_SIZE_Y % THREADS_PER_BLOCK == 0);

    constexpr dim3 block_dim(TILE_SIZE_X, TILE_SIZE_Y);
    const dim3 grid_dim(ceil_div(4096, TILE_SIZE_X), ceil_div(4096, TILE_SIZE_Y));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float total{0};

    for (int i{0}; i < 371; ++i) {
        float milliseconds = 0;
        hipEventRecord(start);
        gemm_2DBT<
            TILE_SIZE_X,
            TILE_SIZE_Y,
            TILE_SIZE_K><<<grid_dim, block_dim>>>(
            dev_A,
            dev_B,
            dev_C,
            4096,
            4096,
            4096,
            4096,
            4096,
            4096
        );
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);

        if (i > 185) total += milliseconds;
        flush_l2_cache();
    }

    std::cout << total / 185 << "\n";

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    delete []host_A;
    delete []host_B;

    flush_l2_cache();
}

void time_gemm_2DBT_async() {
    auto host_A{new float[4096 * 4096]};
    auto host_B{new float[4096 * 4096]};

    float *dev_A;
    float *dev_B;
    float *dev_C;

    fill_matrix(host_A, 4096, 4096, 4096, -100.f, 100.f);
    fill_matrix(host_B, 4096, 4096, 4096, -100.f, 100.f);

    hipMalloc(&dev_A, 4096 * 4096 * sizeof(float));
    hipMalloc(&dev_B, 4096 * 4096 * sizeof(float));
    hipMalloc(&dev_C, 4096 * 4096 * sizeof(float));

    hipMemcpy(dev_A, host_A, 4096 * 4096 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, host_B, 4096 * 4096 * sizeof(float), hipMemcpyHostToDevice);

    constexpr size_t TILE_SIZE_X{16};
    constexpr size_t TILE_SIZE_Y{16};
    constexpr size_t TILE_SIZE_K{16};

    constexpr size_t THREADS_PER_BLOCK{TILE_SIZE_X * TILE_SIZE_Y};

    static_assert(TILE_SIZE_K * TILE_SIZE_X % THREADS_PER_BLOCK == 0);
    static_assert(TILE_SIZE_K * TILE_SIZE_Y % THREADS_PER_BLOCK == 0);

    constexpr dim3 block_dim(TILE_SIZE_X, TILE_SIZE_Y);
    const dim3 grid_dim(ceil_div(4096, TILE_SIZE_X), ceil_div(4096, TILE_SIZE_Y));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float total{0};

    for (int i{0}; i < 371; ++i) {
        float milliseconds = 0;
        hipEventRecord(start);
        gemm_2DBT_async<
            TILE_SIZE_X,
            TILE_SIZE_Y,
            TILE_SIZE_K><<<grid_dim, block_dim>>>(
            dev_A,
            dev_B,
            dev_C,
            4096,
            4096,
            4096,
            4096,
            4096,
            4096
        );
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);

        if (i > 185) total += milliseconds;
        flush_l2_cache();
    }

    std::cout << total / 185 << "\n";

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    delete []host_A;
    delete []host_B;

    flush_l2_cache();
}

void time_gemm_2DBT_2DWT_2DTT_vload() {
    auto host_A{new float[4096 * 4096]};
    auto host_B{new float[4096 * 4096]};

    float *dev_A;
    float *dev_B;
    float *dev_C;

    fill_matrix(host_A, 4096, 4096, 4096, -100.f, 100.f);
    fill_matrix(host_B, 4096, 4096, 4096, -100.f, 100.f);

    hipMalloc(&dev_A, 4096 * 4096 * sizeof(float));
    hipMalloc(&dev_B, 4096 * 4096 * sizeof(float));
    hipMalloc(&dev_C, 4096 * 4096 * sizeof(float));

    hipMemcpy(dev_A, host_A, 4096 * 4096 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, host_B, 4096 * 4096 * sizeof(float), hipMemcpyHostToDevice);

    constexpr uint BLOCK_TILE_SIZE_X{128};
    constexpr uint BLOCK_TILE_SIZE_Y{128};
    constexpr uint BLOCK_TILE_SIZE_K{16};

    constexpr unsigned int WARP_TILE_SIZE_X{64};
    constexpr unsigned int WARP_TILE_SIZE_Y{64};

    constexpr size_t NUM_WARPS_PER_BLOCK_X{BLOCK_TILE_SIZE_X / WARP_TILE_SIZE_X};
    constexpr size_t NUM_WARPS_PER_BLOCK_Y{BLOCK_TILE_SIZE_Y / WARP_TILE_SIZE_Y};

    static_assert(BLOCK_TILE_SIZE_X % WARP_TILE_SIZE_X == 0);
    static_assert(BLOCK_TILE_SIZE_Y % WARP_TILE_SIZE_Y == 0);

    // The size of the internal register caches
    constexpr uint THREAD_TILE_SIZE_Y{8};
    constexpr uint THREAD_TILE_SIZE_X{8};

    constexpr unsigned int NUM_THREADS_PER_WARP_X{4};
    constexpr unsigned int NUM_THREADS_PER_WARP_Y{8};

    static_assert(NUM_THREADS_PER_WARP_X * NUM_THREADS_PER_WARP_Y == 32);

    // ensure each thread stores the same amount of data in their tiles
    static_assert(WARP_TILE_SIZE_X % (THREAD_TILE_SIZE_X * NUM_THREADS_PER_WARP_X) == 0);
    static_assert(WARP_TILE_SIZE_Y % (THREAD_TILE_SIZE_Y * NUM_THREADS_PER_WARP_Y) == 0);

    const dim3 grid_dim{
        ceil_div(4096, BLOCK_TILE_SIZE_X),
        ceil_div(4096, BLOCK_TILE_SIZE_Y)
    };

    constexpr size_t NUM_THREADS_PER_BLOCK{32 * NUM_WARPS_PER_BLOCK_X * NUM_WARPS_PER_BLOCK_Y};
    constexpr dim3 block_dim(NUM_THREADS_PER_BLOCK);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float total{0};

    for (int i{0}; i < 371; ++i) {
        float milliseconds = 0;
        hipEventRecord(start);

        gemm_2DBT_2DWT_2DTT_vload<
            float,
            BLOCK_TILE_SIZE_X,
            BLOCK_TILE_SIZE_Y,
            BLOCK_TILE_SIZE_K,
            WARP_TILE_SIZE_X,
            WARP_TILE_SIZE_Y,
            THREAD_TILE_SIZE_X,
            THREAD_TILE_SIZE_Y,
            NUM_THREADS_PER_WARP_X,
            NUM_THREADS_PER_WARP_Y><<<grid_dim, block_dim>>>(
            dev_A,
            dev_B,
            dev_C,
            1,
            1,
            4096,
            4096,
            4096,
            4096,
            4096,
            4096
        );

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);

        if (i > 185) total += milliseconds;
        flush_l2_cache();
    }

    std::cout << total / 185 << "\n";

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    delete []host_A;
    delete []host_B;

    flush_l2_cache();
}

void time_gemm_2DBT_2DWT_2DTT_async() {
    auto host_A{new float[4096 * 4096]};
    auto host_B{new float[4096 * 4096]};

    float *dev_A;
    float *dev_B;
    float *dev_C;

    fill_matrix(host_A, 4096, 4096, 4096, -100.f, 100.f);
    fill_matrix(host_B, 4096, 4096, 4096, -100.f, 100.f);

    hipMalloc(&dev_A, 4096 * 4096 * sizeof(float));
    hipMalloc(&dev_B, 4096 * 4096 * sizeof(float));
    hipMalloc(&dev_C, 4096 * 4096 * sizeof(float));

    hipMemcpy(dev_A, host_A, 4096 * 4096 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, host_B, 4096 * 4096 * sizeof(float), hipMemcpyHostToDevice);

    constexpr uint BLOCK_TILE_SIZE_X{128};
    constexpr uint BLOCK_TILE_SIZE_Y{128};
    constexpr uint BLOCK_TILE_SIZE_K{16};

    constexpr unsigned int WARP_TILE_SIZE_X{64};
    constexpr unsigned int WARP_TILE_SIZE_Y{64};

    constexpr size_t NUM_WARPS_PER_BLOCK_X{BLOCK_TILE_SIZE_X / WARP_TILE_SIZE_X};
    constexpr size_t NUM_WARPS_PER_BLOCK_Y{BLOCK_TILE_SIZE_Y / WARP_TILE_SIZE_Y};

    static_assert(BLOCK_TILE_SIZE_X % WARP_TILE_SIZE_X == 0);
    static_assert(BLOCK_TILE_SIZE_Y % WARP_TILE_SIZE_Y == 0);

    // The size of the internal register caches
    constexpr uint THREAD_TILE_SIZE_Y{8};
    constexpr uint THREAD_TILE_SIZE_X{8};

    constexpr unsigned int NUM_THREADS_PER_WARP_X{4};
    constexpr unsigned int NUM_THREADS_PER_WARP_Y{8};

    static_assert(NUM_THREADS_PER_WARP_X * NUM_THREADS_PER_WARP_Y == 32);

    // ensure each thread stores the same amount of data in their tiles
    static_assert(WARP_TILE_SIZE_X % (THREAD_TILE_SIZE_X * NUM_THREADS_PER_WARP_X) == 0);
    static_assert(WARP_TILE_SIZE_Y % (THREAD_TILE_SIZE_Y * NUM_THREADS_PER_WARP_Y) == 0);

    const dim3 grid_dim{
        ceil_div(4096, BLOCK_TILE_SIZE_X),
        ceil_div(4096, BLOCK_TILE_SIZE_Y)
    };

    constexpr size_t NUM_THREADS_PER_BLOCK{32 * NUM_WARPS_PER_BLOCK_X * NUM_WARPS_PER_BLOCK_Y};
    constexpr dim3 block_dim(NUM_THREADS_PER_BLOCK);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float total{0};

    for (int i{0}; i < 371; ++i) {
        float milliseconds = 0;
        hipEventRecord(start);

        gemm_2DBT_2DWT_2DTT_async_load<
            float,
            BLOCK_TILE_SIZE_X,
            BLOCK_TILE_SIZE_Y,
            BLOCK_TILE_SIZE_K,
            WARP_TILE_SIZE_X,
            WARP_TILE_SIZE_Y,
            THREAD_TILE_SIZE_X,
            THREAD_TILE_SIZE_Y,
            NUM_THREADS_PER_WARP_X,
            NUM_THREADS_PER_WARP_Y><<<grid_dim, block_dim>>>(
            dev_A,
            dev_B,
            dev_C,
            1,
            1,
            4096,
            4096,
            4096,
            4096,
            4096,
            4096
        );

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);

        if (i > 185) total += milliseconds;
        flush_l2_cache();
    }

    std::cout << total / 185 << "\n";

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    delete []host_A;
    delete []host_B;

    flush_l2_cache();
}

void time_db_gemm_memory() {
    auto host_A{new float[4096 * 4096]};
    auto host_B{new float[4096 * 4096]};

    float *dev_A;
    float *dev_B;
    float *dev_C;

    fill_matrix(host_A, 4096, 4096, 4096, -100.f, 100.f);
    fill_matrix(host_B, 4096, 4096, 4096, -100.f, 100.f);

    hipMalloc(&dev_A, 4096 * 4096 * sizeof(float));
    hipMalloc(&dev_B, 4096 * 4096 * sizeof(float));
    hipMalloc(&dev_C, 4096 * 4096 * sizeof(float));

    hipMemcpy(dev_A, host_A, 4096 * 4096 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, host_B, 4096 * 4096 * sizeof(float), hipMemcpyHostToDevice);

    constexpr size_t TILE_SIZE_X{32};
    constexpr size_t TILE_SIZE_Y{32};
    constexpr size_t TILE_SIZE_K{32};

    constexpr size_t THREADS_PER_BLOCK{TILE_SIZE_X * TILE_SIZE_Y};

    static_assert(TILE_SIZE_K * TILE_SIZE_X % THREADS_PER_BLOCK == 0);
    static_assert(TILE_SIZE_K * TILE_SIZE_Y % THREADS_PER_BLOCK == 0);

    constexpr dim3 block_dim(TILE_SIZE_X, TILE_SIZE_Y);
    const dim3 grid_dim(ceil_div(4096, TILE_SIZE_X), ceil_div(4096, TILE_SIZE_Y));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float total{0};

    for (int i{0}; i < 371; ++i) {
        float milliseconds = 0;
        hipEventRecord(start);
        gemm_double_buffering<
            THREADS_PER_BLOCK,
            TILE_SIZE_X,
            TILE_SIZE_Y,
            TILE_SIZE_K><<<grid_dim, block_dim>>>(
            dev_A,
            dev_B,
            dev_C,
            4096,
            4096,
            4096,
            4096,
            4096,
            4096
        );
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);

        if (i > 185) total += milliseconds;
        flush_l2_cache();
    }

    std::cout << total / 185 << "\n";

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    delete []host_A;
    delete []host_B;
}

void time_double_buffer() {
    auto host_A{new float[4096 * 4096]};
    auto host_B{new float[4096 * 4096]};

    float *dev_A;
    float *dev_B;
    float *dev_C;

    fill_matrix(host_A, 4096, 4096, 4096, -100.f, 100.f);
    fill_matrix(host_B, 4096, 4096, 4096, -100.f, 100.f);

    hipMalloc(&dev_A, 4096 * 4096 * sizeof(float));
    hipMalloc(&dev_B, 4096 * 4096 * sizeof(float));
    hipMalloc(&dev_C, 4096 * 4096 * sizeof(float));

    hipMemcpy(dev_A, host_A, 4096 * 4096 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, host_B, 4096 * 4096 * sizeof(float), hipMemcpyHostToDevice);

    constexpr size_t TILE_SIZE_X{16};
    constexpr size_t TILE_SIZE_Y{16};
    constexpr size_t TILE_SIZE_K{16};
    constexpr size_t CONSUMER_WARPS{16 * 16 / 32};
    constexpr size_t PRODUCER_WARPS{16 * 16 / 32};

    constexpr size_t THREADS_PER_BLOCK{TILE_SIZE_X * TILE_SIZE_Y};

    static_assert(TILE_SIZE_K * TILE_SIZE_X % THREADS_PER_BLOCK == 0);
    static_assert(TILE_SIZE_K * TILE_SIZE_Y % THREADS_PER_BLOCK == 0);

    constexpr dim3 block_dim(PRODUCER_WARPS * 32 + CONSUMER_WARPS * 32);
    const dim3 grid_dim(ceil_div(4096, TILE_SIZE_X), ceil_div(4096, TILE_SIZE_Y));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    producer_consumer_pattern<
        CONSUMER_WARPS,
        PRODUCER_WARPS,
        32,
        TILE_SIZE_X,
        TILE_SIZE_Y,
        TILE_SIZE_K><<<grid_dim, block_dim>>>(
        dev_A,
        dev_B,
        dev_C,
        4096,
        4096,
        4096,
        4096,
        4096,
        4096
    );
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << milliseconds;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    delete []host_A;
    delete []host_B;
}


void run_double_buffer_test() {
    // test_regular_shared_mem();
    // test_double_buffer_gemm();
    // time_db_gemm_memory();
    // test_gemm_2DBT_async();
    // test_async_load_gemm();
    // time_2DBT();
    // time_gemm_2DBT_async();
    time_gemm_2DBT_2DWT_2DTT_vload();
    time_gemm_2DBT_2DWT_2DTT_async();
    // test_gemm_2DBT_2DWT_2DTT_vload();
    // time_double_buffer();
}
