#include "hip/hip_runtime.h"
//
// Created by sriram on 5/27/25.
//

#include "../../src/gemm/cutlass_gemm.cuh"
#include "../test_helpers.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>


// Regular implementation
template<
    typename A_GLOBAL_LAYOUT,
    typename A_GLOBAL_ENGINE,
    typename A_SHARED_LAYOUT,
    typename A_SHARED_ENGINE,
    typename B_GLOBAL_LAYOUT,
    typename B_GLOBAL_ENGINE,
    typename B_SHARED_LAYOUT,
    typename B_SHARED_ENGINE,
    typename THREAD_LAYOUT
>
CUTE_DEVICE void load_to_shared(
    const cute::Tensor<A_SHARED_ENGINE, A_SHARED_LAYOUT> &shared_A,
    const cute::Tensor<B_SHARED_ENGINE, B_SHARED_LAYOUT> &shared_B,
    const cute::Tensor<A_GLOBAL_ENGINE, A_GLOBAL_LAYOUT> &global_A,
    const cute::Tensor<B_GLOBAL_ENGINE, B_GLOBAL_LAYOUT> &global_B,
    const THREAD_LAYOUT &thread_layout
) {
    using namespace cute;

    constexpr size_t smem_length_A{cosize_v<A_SHARED_LAYOUT>};
    constexpr size_t smem_length_B{cosize_v<B_SHARED_LAYOUT>};
    constexpr size_t thread_length{cosize_v<THREAD_LAYOUT>};

    static_assert(smem_length_A % thread_length == 0);
    static_assert(smem_length_B % thread_length == 0);

    static_assert(size<0>(A_GLOBAL_LAYOUT{}) == size<0>(A_SHARED_LAYOUT{}));
    static_assert(size<1>(A_GLOBAL_LAYOUT{}) == size<1>(A_SHARED_LAYOUT{}));
    static_assert(size<0>(B_GLOBAL_LAYOUT{}) == size<0>(B_SHARED_LAYOUT{}));
    static_assert(size<1>(B_GLOBAL_LAYOUT{}) == size<1>(B_SHARED_LAYOUT{}));

    constexpr size_t A_loads_per_thread{smem_length_A / thread_length};
    constexpr size_t B_loads_per_thread{smem_length_B / thread_length};

    constexpr auto tv_layout_A{
        make_layout(
            make_shape(make_shape(size<1>(A_GLOBAL_LAYOUT{}),
                                  size<0>(A_GLOBAL_LAYOUT{}) / A_loads_per_thread), A_loads_per_thread),
            make_stride(make_stride(size<0>(A_GLOBAL_LAYOUT{}), A_loads_per_thread), _1{}))
    };

    constexpr auto tv_layout_B{
        make_layout(
            make_shape(
                make_shape(size<1>(B_GLOBAL_LAYOUT{}), size<0>(B_GLOBAL_LAYOUT{}) / B_loads_per_thread),
                B_loads_per_thread),
            make_stride(make_stride(size<0>(B_GLOBAL_LAYOUT{}), B_loads_per_thread), _1{}))
    };

    Tensor shared_A_tv{composition(shared_A, tv_layout_A)};
    Tensor shared_B_tv{composition(shared_B, tv_layout_B)};
    const Tensor global_A_tv{composition(global_A, tv_layout_A)};
    const Tensor global_B_tv{composition(global_B, tv_layout_B)};

    const Tensor global_A_value{global_A_tv(threadIdx.x, _)};
    Tensor shared_A_value{shared_A_tv(threadIdx.x, _)};

    const Tensor global_B_value{global_B_tv(threadIdx.x, _)};
    Tensor shared_B_value{shared_B_tv(threadIdx.x, _)};

    copy(global_A_value, shared_A_value);
    copy(global_B_value, shared_B_value);
}


template<
    typename A_GLOBAL_LAYOUT,
    typename A_GLOBAL_ENGINE,
    typename A_SHARED_T_LAYOUT,
    typename A_SHARED_T_ENGINE,
    typename B_GLOBAL_LAYOUT,
    typename B_GLOBAL_ENGINE,
    typename B_SHARED_LAYOUT,
    typename B_SHARED_ENGINE,
    typename THREAD_LAYOUT
>
CUTE_DEVICE void load_to_shared_transposed(
    const cute::Tensor<A_SHARED_T_ENGINE, A_SHARED_T_LAYOUT> &shared_A_transposed,
    const cute::Tensor<B_SHARED_ENGINE, B_SHARED_LAYOUT> &shared_B,
    const cute::Tensor<A_GLOBAL_ENGINE, A_GLOBAL_LAYOUT> &global_A,
    const cute::Tensor<B_GLOBAL_ENGINE, B_GLOBAL_LAYOUT> &global_B,
    const THREAD_LAYOUT &thread_layout
) {
    using namespace cute;

    constexpr size_t smem_length_A{cosize_v<A_SHARED_T_LAYOUT>};
    constexpr size_t smem_length_B{cosize_v<B_SHARED_LAYOUT>};
    constexpr size_t thread_length{cosize_v<THREAD_LAYOUT>};

    static_assert(smem_length_A % thread_length == 0);
    static_assert(smem_length_B % thread_length == 0);
    static_assert(size<0>(A_GLOBAL_LAYOUT{}) == size<1>(A_SHARED_T_LAYOUT{}));
    static_assert(size<1>(A_GLOBAL_LAYOUT{}) == size<0>(A_SHARED_T_LAYOUT{}));
    static_assert(size<0>(B_GLOBAL_LAYOUT{}) == size<0>(B_SHARED_LAYOUT{}));
    static_assert(size<1>(B_GLOBAL_LAYOUT{}) == size<1>(B_SHARED_LAYOUT{}));

    constexpr size_t A_loads_per_thread{smem_length_A / thread_length};
    constexpr size_t B_loads_per_thread{smem_length_B / thread_length};

    constexpr auto tv_layout_A{
        make_layout(
            make_shape(make_shape(size<1>(A_GLOBAL_LAYOUT{}),
                                  size<0>(A_GLOBAL_LAYOUT{}) / A_loads_per_thread), A_loads_per_thread),
            make_stride(make_stride(size<0>(A_GLOBAL_LAYOUT{}), A_loads_per_thread), _1{}))
    };

    constexpr auto tv_layout_B{
        make_layout(
            make_shape(
                make_shape(size<1>(B_GLOBAL_LAYOUT{}), size<0>(B_GLOBAL_LAYOUT{}) / B_loads_per_thread),
                B_loads_per_thread),
            make_stride(make_stride(size<0>(B_GLOBAL_LAYOUT{}), B_loads_per_thread), _1{}))
    };

    Tensor shared_A_tv{composition(shared_A_transposed, tv_layout_A)};
    Tensor shared_B_tv{composition(shared_B, tv_layout_B)};
    const Tensor global_A_tv{composition(global_A, tv_layout_A)};
    const Tensor global_B_tv{composition(global_B, tv_layout_B)};

    const Tensor global_A_value{global_A_tv(threadIdx.x, _)};
    Tensor shared_A_value{shared_A_tv(threadIdx.x, _)};

    const Tensor global_B_value{global_B_tv(threadIdx.x, _)};
    Tensor shared_B_value{shared_B_tv(threadIdx.x, _)};

    copy(global_A_value, shared_A_value);
    copy(global_B_value, shared_B_value);
}

template<
    typename T,
    typename A_GLOBAL_LAYOUT,
    typename A_SHARED_LAYOUT,
    typename B_GLOBAL_LAYOUT,
    typename B_SHARED_LAYOUT,
    typename C_GLOBAL_LAYOUT,
    typename THREAD_LAYOUT
>
__global__ static void gemm_2DBT(
    const T *gmem_A,
    const T *gmem_B,
    T *gmem_C,
    const A_GLOBAL_LAYOUT gmem_layout_A,
    const B_GLOBAL_LAYOUT gmem_layout_B,
    const C_GLOBAL_LAYOUT gmem_layout_C,
    const A_SHARED_LAYOUT smem_layout_A,
    const B_SHARED_LAYOUT smem_layout_B,
    const THREAD_LAYOUT thread_layout,
    const T alpha,
    const T beta
) {
    using namespace cute;
    static_assert(gmem_layout_A.rank == 2);
    static_assert(gmem_layout_B.rank == 2);
    static_assert(gmem_layout_C.rank == 2);
    static_assert(smem_layout_A.rank == 2);
    static_assert(smem_layout_B.rank == 2);

    constexpr size_t BLOCK_TILE_SIZE_K{size<1>(smem_layout_A)};
    static_assert(BLOCK_TILE_SIZE_K == size<0>(smem_layout_B));

    extern __shared__ T shared_memory[];
    constexpr size_t smem_length_A{cosize_v<A_SHARED_LAYOUT>};

    // helps with deciding what copy algorithm to use
    smem_ptr pShared_A{make_smem_ptr(shared_memory)};
    smem_ptr pShared_B{make_smem_ptr(&shared_memory[smem_length_A])};
    gmem_ptr pGlobal_A{make_gmem_ptr(gmem_A)};
    gmem_ptr pGlobal_B{make_gmem_ptr(gmem_B)};
    gmem_ptr pGlobal_C{make_gmem_ptr(gmem_C)};

    Tensor shared_A{make_tensor(pShared_A, smem_layout_A)};
    Tensor shared_B{make_tensor(pShared_B, smem_layout_B)};
    Tensor global_A{make_tensor(pGlobal_A, gmem_layout_A)};
    Tensor global_B{make_tensor(pGlobal_B, gmem_layout_B)};
    Tensor global_C{make_tensor(pGlobal_C, gmem_layout_C)};

    const size_t total_iters{ceil_div(size<1>(gmem_layout_A), BLOCK_TILE_SIZE_K)};

    Tensor gA_tiled{zipped_divide(global_A, smem_layout_A.shape())};
    Tensor gB_tiled{zipped_divide(global_B, smem_layout_B.shape())};
    Tensor gC_tiled{zipped_divide(global_C, thread_layout.shape())};

    Tensor tile_C{gC_tiled(make_coord(_, _), make_coord(blockIdx.y, blockIdx.x))};

    T partial{0};

    auto coords{idx2crd(threadIdx.x, thread_layout.shape(), thread_layout.stride())};
    const size_t row{coords.first_};
    const size_t col{coords.rest_.first_};

    for (size_t iter{0}; iter < total_iters; ++iter) {
        Tensor tile_A{gA_tiled(make_coord(_, _), make_coord(blockIdx.y, iter))};
        Tensor tile_B{gB_tiled(make_coord(_, _), make_coord(iter, blockIdx.x))};

        // load to shared
        load_to_shared(shared_A, shared_B, tile_A, tile_B, thread_layout);
        __syncthreads();

        Tensor slice_A{shared_A(make_coord(row, _))};
        Tensor slice_B{shared_B(make_coord(_, col))};

#pragma unroll
        for (size_t kk{0}; kk < BLOCK_TILE_SIZE_K; ++kk) {
            partial += slice_A(kk) * slice_B(kk);
        }
        __syncthreads();
    }

    tile_C(make_coord(row, col)) = tile_C(make_coord(row, col)) * beta + partial * alpha;
}

void test_cute_gemm_2DBT() {
    using namespace cute;

    constexpr size_t M{128};
    constexpr size_t N{64};
    constexpr size_t K{256};

    constexpr size_t BLOCK_TILE_SIZE_Y{16};
    constexpr size_t BLOCK_TILE_SIZE_X{16};
    constexpr size_t BLOCK_TILE_SIZE_K{16};

    static_assert((M * K) % (BLOCK_TILE_SIZE_Y * BLOCK_TILE_SIZE_K) == 0);
    static_assert((N * K) % (BLOCK_TILE_SIZE_K * BLOCK_TILE_SIZE_X) == 0);

    thrust::host_vector<float> host_matrixA(M * K);
    thrust::host_vector<float> host_matrixB(K * N);
    thrust::host_vector<float> host_matrixC(M * N);

    fill_matrix_w(host_matrixA.data(), M, K, K, -100, 100);
    fill_matrix_w(host_matrixB.data(), K, N, N, -100, 100);

    for (size_t i{0}; i < N * M; ++i) host_matrixC[i] = 0.f;

    thrust::device_vector<float> device_matrixA{host_matrixA};
    thrust::device_vector<float> device_matrixB{host_matrixB};
    thrust::device_vector<float> device_matrixC{host_matrixC};

    const Layout gmem_A_lo{make_layout(make_shape(M, K), LayoutRight{})};
    const Layout gmem_B_lo{make_layout(make_shape(K, N), LayoutRight{})};
    const Layout gmem_C_lo{make_layout(make_shape(M, N), LayoutRight{})};

    // print2D_tensor(make_tensor(host_matrixA.data(), gmem_A_lo));

    constexpr Layout smem_A_lo{
        make_layout(make_shape(Int<BLOCK_TILE_SIZE_Y>{}, Int<BLOCK_TILE_SIZE_K>{}), LayoutRight{})
    };
    constexpr Layout smem_B_lo{
        make_layout(make_shape(Int<BLOCK_TILE_SIZE_K>{}, Int<BLOCK_TILE_SIZE_X>{}), LayoutRight{})
    };
    constexpr Layout thread_lo{
        make_layout(make_shape(Int<16>{}, Int<16>{}), LayoutRight{})
    };

    constexpr size_t shared_mem_size{
        (BLOCK_TILE_SIZE_Y * BLOCK_TILE_SIZE_K) + (BLOCK_TILE_SIZE_K * BLOCK_TILE_SIZE_X) * sizeof(float)
    };

    dim3 grid_dim{
        ceil_div(N, BLOCK_TILE_SIZE_X),
        ceil_div(M, BLOCK_TILE_SIZE_Y)
    };

    dim3 block_dim{
        BLOCK_TILE_SIZE_X * BLOCK_TILE_SIZE_Y
    };

    gemm_2DBT<<<grid_dim, block_dim, shared_mem_size>>>(
        device_matrixA.data().get(),
        device_matrixB.data().get(),
        device_matrixC.data().get(),
        gmem_A_lo,
        gmem_B_lo,
        gmem_C_lo,
        smem_A_lo,
        smem_B_lo,
        thread_lo,
        1.f,
        1.f);

    thrust::host_vector<float> host_matrixC2{device_matrixC};
    cpu_matmul_naive(host_matrixA.data(), host_matrixB.data(), host_matrixC.data(), M, N, K, K, N, N);
    test_equivalency(host_matrixC.data(), host_matrixC2.data(), M, N, N);
}

template<typename TensorLayout, typename TensorEngine>
void print_ten(cute::Tensor<TensorEngine, TensorLayout> tens) {
    // cute::print_layout(tens.layout());

    for (size_t m{0}; m < cute::size<0>(tens); ++m) {
        std::cout << "[ ";
        for (size_t n{0}; n < cute::size<1>(tens); ++n) {
            std::cout << tens(cute::make_coord(m, n)) << " ";
        }
        std::cout << "]\n";
    }
}

template<
    typename T,
    typename A_GLOBAL_LAYOUT,
    typename B_GLOBAL_LAYOUT,
    typename C_GLOBAL_LAYOUT,
    size_t BLOCK_TILE_SIZE_X,
    size_t BLOCK_TILE_SIZE_Y,
    size_t BLOCK_TILE_SIZE_K,
    size_t WARP_TILE_SIZE_X,
    size_t WARP_TILE_SIZE_Y,
    size_t THREAD_TILE_SIZE_X,
    size_t THREAD_TILE_SIZE_Y,
    size_t NUM_THREADS_PER_WARP_X,
    size_t NUM_THREADS_PER_WARP_Y
>
__global__ static void gemm_2DBT_2DWT_2DTT_vloadT(
    const T *gmem_A,
    const T *gmem_B,
    T *gmem_C,
    const A_GLOBAL_LAYOUT gmem_layout_A,
    const B_GLOBAL_LAYOUT gmem_layout_B,
    const C_GLOBAL_LAYOUT gmem_layout_C,
    const T alpha,
    const T beta
) {
    using namespace cute;

    constexpr size_t NUM_WARPS_X{BLOCK_TILE_SIZE_X / WARP_TILE_SIZE_X};
    constexpr size_t NUM_WARPS_Y{BLOCK_TILE_SIZE_Y / WARP_TILE_SIZE_Y};

    constexpr size_t NUM_CACHES_PER_WARP_X{WARP_TILE_SIZE_X / (THREAD_TILE_SIZE_X * NUM_THREADS_PER_WARP_X)};
    constexpr size_t NUM_CACHES_PER_WARP_Y{WARP_TILE_SIZE_Y / (THREAD_TILE_SIZE_Y * NUM_THREADS_PER_WARP_Y)};

    static_assert(gmem_layout_A.rank == 2);
    static_assert(gmem_layout_B.rank == 2);
    static_assert(gmem_layout_C.rank == 2);

    static_assert(size<1>(A_GLOBAL_LAYOUT{}) == size<0>(B_GLOBAL_LAYOUT{}));
    // TODO check c as well

    // Divides each thread into a warp block where further values can then be derivec
    constexpr Layout thread_layout{
        make_layout(
            make_shape(
                make_shape(Int<NUM_THREADS_PER_WARP_Y>{}, Int<NUM_WARPS_Y>{}),
                make_shape(Int<NUM_THREADS_PER_WARP_X>{}, Int<NUM_WARPS_X>{})),
            make_stride(
                make_stride(Int<NUM_THREADS_PER_WARP_X>{}, Int<NUM_WARPS_X * 32>{}),
                make_stride(_1{}, _32{})
            )
        )
    };

    constexpr Layout smem_A_T_layout{
        make_layout(make_shape(Int<BLOCK_TILE_SIZE_K>{}, Int<BLOCK_TILE_SIZE_Y>{}), LayoutRight{})
    };
    constexpr Layout smem_B_layout{
        make_layout(make_shape(Int<BLOCK_TILE_SIZE_K>{}, Int<BLOCK_TILE_SIZE_X>{}), LayoutRight{})
    };

    constexpr auto warp_tile{
        make_shape(Int<WARP_TILE_SIZE_Y>{}, Int<WARP_TILE_SIZE_X>{})
    };

    constexpr auto A_block_tiler{
        make_shape(Int<BLOCK_TILE_SIZE_Y>{}, Int<BLOCK_TILE_SIZE_K>{})
    };

    constexpr auto C_block_tiler{
        make_shape(Int<BLOCK_TILE_SIZE_Y>{}, Int<BLOCK_TILE_SIZE_X>{})
    };

    constexpr auto warp_slice_tile_A{
        make_layout(
            make_shape(
                make_shape(Int<NUM_THREADS_PER_WARP_Y>{}, Int<NUM_CACHES_PER_WARP_Y>{}),
                make_shape(Int<THREAD_TILE_SIZE_Y>{})
            ),
            make_stride(
                make_stride(Int<THREAD_TILE_SIZE_Y>{}, Int<THREAD_TILE_SIZE_Y * NUM_THREADS_PER_WARP_Y>{}),
                make_stride(_1{})
            )
        )
    };

    constexpr auto warp_slice_tile_B{
        make_layout(
            make_shape(
                make_shape(Int<NUM_THREADS_PER_WARP_X>{}, Int<NUM_CACHES_PER_WARP_X>{}),
                Int<THREAD_TILE_SIZE_X>{}
            ),
            make_stride(
                make_stride(Int<THREAD_TILE_SIZE_X>{}, Int<THREAD_TILE_SIZE_X * NUM_THREADS_PER_WARP_X>{}),
                _1{}
            )
        )
    };

    // This layout maps an entire warp tile to all 32 threads it assigns the threads
    // THREAD_TILE_SIZE_Y * THREAD_TILE_SIZE_X, matrices since that is what each thread is supposed to compute
    // however in the scenario where threads are responsible to compute multiple tiles (NUM_CACHES_PER_WARP) is greater
    // than 1 then we also group those matrices near each other for easy indexing
    //
    // the layout is organized as a high dimensional matrix. The first dimension is THREAD_TILE_SIZE_Y, as this is the
    // row length of the matrices, this gets repeated NUM_CACHES_PER_WARP_X * NUM_CACHES_PER_WARP_Y times since that
    // rows of matrices are being computed, the next coord pair correlate the cluster of matrices to the appropriate
    // warp idx pair
    // Finally the last idx specifies the length of each row which is THREAD_TILE_SIZE_X
    // To access the first cache matrix for warp index 0,2 this is the command
    // tv(make_coord(make_coord(_, make_coord(make_coord(0, 0), make_coord(0, 2))), _))
    // TODO: Add visual description
    constexpr Layout warp_tv_layout{
        make_layout(
            make_shape(
                make_shape(
                    Int<THREAD_TILE_SIZE_Y>{},
                    make_shape(
                        make_shape(Int<NUM_CACHES_PER_WARP_Y>{}, Int<NUM_CACHES_PER_WARP_X>{}),
                        make_shape(Int<NUM_THREADS_PER_WARP_Y>{}, Int<NUM_THREADS_PER_WARP_X>{})
                    )
                ),
                Int<THREAD_TILE_SIZE_X>{}
            ),
            make_stride(
                make_stride(
                    _1{},
                    make_stride(
                        make_stride(
                            Int<THREAD_TILE_SIZE_Y * NUM_THREADS_PER_WARP_Y>{},
                            Int<THREAD_TILE_SIZE_X * NUM_THREADS_PER_WARP_X * WARP_TILE_SIZE_Y>{}
                        ),
                        make_stride(Int<THREAD_TILE_SIZE_Y>{}, Int<THREAD_TILE_SIZE_X * WARP_TILE_SIZE_Y>{}))),
                Int<WARP_TILE_SIZE_Y>{}
            )
        )
    };

    extern __shared__ T shared_memory[];
    constexpr size_t smem_length_A{cosize_v<decltype(smem_A_T_layout)>};

    // helps with deciding what copy algorithm to use
    smem_ptr pShared_A{make_smem_ptr(shared_memory)};
    smem_ptr pShared_B{make_smem_ptr(&shared_memory[smem_length_A])};
    gmem_ptr pGlobal_A{make_gmem_ptr(gmem_A)};
    gmem_ptr pGlobal_B{make_gmem_ptr(gmem_B)};
    gmem_ptr pGlobal_C{make_gmem_ptr(gmem_C)};

    Tensor shared_A{make_tensor(pShared_A, smem_A_T_layout)};
    Tensor shared_B{make_tensor(pShared_B, smem_B_layout)};
    Tensor global_A{make_tensor(pGlobal_A, gmem_layout_A)};
    Tensor global_B{make_tensor(pGlobal_B, gmem_layout_B)};
    Tensor global_C{make_tensor(pGlobal_C, gmem_layout_C)};

    const size_t total_iters{ceil_div(size<1>(gmem_layout_A), BLOCK_TILE_SIZE_K)};

    const auto warp_coord{idx2crd(threadIdx.x, thread_layout.shape(), thread_layout.stride())};
    const size_t row_in_warp{warp_coord.first_.first_}; // index of the row in the warp_tile
    const size_t warp_y_idx{warp_coord.first_.rest_.first_}; // index of the warp_tile in the block
    const size_t col_in_warp{warp_coord.rest_.first_.first_}; // index of the col in the warp_tile
    const size_t warp_x_idx{warp_coord.rest_.first_.rest_.first_}; // index of the warp_tile in the block


     Tensor block_tile_C{local_tile(global_C, C_block_tiler, make_coord(blockIdx.y, blockIdx.x))};
     Tensor warp_tile_C{local_tile(block_tile_C, warp_tile, make_coord(warp_y_idx, warp_x_idx))};
     Tensor tv_warp_tile_C{composition(warp_tile_C, warp_tv_layout)};
     Tensor C_value{
         tv_warp_tile_C(make_coord(make_coord(_, make_coord(make_coord(_, _), make_coord(row_in_warp, col_in_warp))), _))
     };

     Tensor rmem_A_cache{
         make_tensor<T>(
             make_shape(Int<NUM_CACHES_PER_WARP_Y>(), Int<THREAD_TILE_SIZE_Y>()),
             LayoutRight{}
         )
     };

     Tensor rmem_B_cache{
         make_tensor<T>(
             make_shape(Int<NUM_CACHES_PER_WARP_X>(), Int<THREAD_TILE_SIZE_X>()),
             LayoutRight{}
         )
     };

     Tensor rmem_cache_intermediates{
         make_tensor<T>(
             make_shape(
                 make_shape(
                     Int<THREAD_TILE_SIZE_Y>{},
                     make_shape(Int<NUM_CACHES_PER_WARP_Y>{}, Int<NUM_CACHES_PER_WARP_X>{})
                 ),
                 Int<THREAD_TILE_SIZE_X>{}
             ),
             make_stride(
                 make_stride(
                     Int<THREAD_TILE_SIZE_X * NUM_CACHES_PER_WARP_X>{},
                     make_stride(Int<THREAD_TILE_SIZE_X * NUM_CACHES_PER_WARP_X * THREAD_TILE_SIZE_Y>{},
                                 Int<THREAD_TILE_SIZE_X>{})
                 ),
                 _1{}
             )
         )
     };

     for (size_t iter{0}; iter < total_iters; ++iter) {
         // same as zipped divide w/ less code
         Tensor tile_A{local_tile(global_A, A_block_tiler, make_coord(blockIdx.y, iter))};
         Tensor tile_B{local_tile(global_B, smem_B_layout.shape(), make_coord(iter, blockIdx.x))};

         // load to shared
         // TODO change
         // load_to_shared(shared_A, shared_B, tile_A, tile_B, thread_layout);
         __syncthreads();

         Tensor warp_tile_A{
             local_tile(shared_A, make_shape(Int<BLOCK_TILE_SIZE_K>{}, Int<WARP_TILE_SIZE_Y>{}),
                        make_coord(0, warp_y_idx))
         };
         Tensor warp_tile_B{
             local_tile(shared_B, make_shape(Int<BLOCK_TILE_SIZE_K>{}, Int<WARP_TILE_SIZE_X>{}),
                        make_coord(0, warp_x_idx))
         };

#pragma unroll
        for (size_t kk{0}; kk < BLOCK_TILE_SIZE_K; ++kk) {
            Tensor slice_warp_tile_A{warp_tile_A(make_coord(kk, _))};
            Tensor slice_warp_tile_B{warp_tile_B(make_coord(kk, _))};

            Tensor slice_warp_A_tv{composition(slice_warp_tile_A, warp_slice_tile_A)};
            Tensor slice_warp_B_tv{composition(slice_warp_tile_B, warp_slice_tile_B)};

            copy(rmem_A_cache, slice_warp_A_tv(make_coord(make_coord(row_in_warp, _), _)));
            copy(rmem_B_cache, slice_warp_B_tv(make_coord(make_coord(col_in_warp, _), _)));

            for (size_t rmem_cache_idxA{0}; rmem_cache_idxA < NUM_CACHES_PER_WARP_Y; ++rmem_cache_idxA) {
                for (size_t rmem_cache_idxB{0}; rmem_cache_idxB < NUM_CACHES_PER_WARP_X; ++rmem_cache_idxB) {
                    Tensor A_cache_slice{rmem_A_cache(make_coord(_, rmem_cache_idxA))};
                    Tensor B_cache_slice{rmem_B_cache(make_coord(_, rmem_cache_idxB))};
                    Tensor partials{
                        rmem_cache_intermediates(make_coord(make_coord(_, make_coord(rmem_cache_idxA, rmem_cache_idxB)), _))
                    };

                    for (size_t i{0}; i < THREAD_TILE_SIZE_Y; ++i) {
                        T acs{A_cache_slice(i)};
                        for (size_t j{0}; j < THREAD_TILE_SIZE_X; ++j) {
                            partials(make_coord(i, j)) = acs * B_cache_slice(j);
                        }
                    }
                }
            }
        }
         __syncthreads();
     }

     axpby(alpha, rmem_cache_intermediates, beta, C_value);
}


void test_cute_gemm_2DBT_2DWT_2DTT_vloadT() {
    using namespace cute;

    constexpr size_t M{128};
    constexpr size_t N{128};
    constexpr size_t K{256};

    constexpr size_t BLOCK_TILE_SIZE_X{128};
    constexpr size_t BLOCK_TILE_SIZE_Y{128};
    constexpr size_t BLOCK_TILE_SIZE_K{16};

    constexpr size_t WARP_TILE_SIZE_X{32};
    constexpr size_t WARP_TILE_SIZE_Y{64};

    static_assert((M * K) % (BLOCK_TILE_SIZE_Y * BLOCK_TILE_SIZE_K) == 0);
    static_assert((N * K) % (BLOCK_TILE_SIZE_K * BLOCK_TILE_SIZE_X) == 0);

    constexpr size_t NUM_WARPS_X{BLOCK_TILE_SIZE_X / WARP_TILE_SIZE_X};
    constexpr size_t NUM_WARPS_Y{BLOCK_TILE_SIZE_Y / WARP_TILE_SIZE_Y};

    static_assert(BLOCK_TILE_SIZE_X % WARP_TILE_SIZE_X == 0);
    static_assert(BLOCK_TILE_SIZE_Y % WARP_TILE_SIZE_Y == 0);

    constexpr size_t THREAD_TILE_SIZE_X{8};
    constexpr size_t THREAD_TILE_SIZE_Y{8};

    constexpr size_t NUM_THREADS_PER_WARP_X{4};
    constexpr size_t NUM_THREADS_PER_WARP_Y{8};

    static_assert(NUM_THREADS_PER_WARP_X * NUM_THREADS_PER_WARP_Y == 32);

    static_assert(WARP_TILE_SIZE_X % (THREAD_TILE_SIZE_X * NUM_THREADS_PER_WARP_X) == 0);
    static_assert(WARP_TILE_SIZE_Y % (THREAD_TILE_SIZE_Y * NUM_THREADS_PER_WARP_Y) == 0);

    constexpr size_t THREADS_PER_BLOCK{32 * NUM_WARPS_X * NUM_WARPS_Y};

    thrust::host_vector<float> host_matrixA(M * K);
    thrust::host_vector<float> host_matrixB(K * N);
    thrust::host_vector<float> host_matrixC(M * N);

    thrust::device_vector<float> d_matrixA{host_matrixA};
    thrust::device_vector<float> d_matrixB{host_matrixB};
    thrust::device_vector<float> d_matrixC{host_matrixC};

    fill_matrix_w(host_matrixA.data(), M, K, K, -100, 100);
    fill_matrix_w(host_matrixB.data(), K, N, N, -100, 100);
    for (size_t i{0}; i < N * M; ++i) host_matrixC[i] = 0.f;

    dim3 grid_dim{
        ceil_div(N, BLOCK_TILE_SIZE_X),
        ceil_div(M, BLOCK_TILE_SIZE_Y)
    };

    gemm_2DBT_2DWT_2DTT_vloadT<
        float,
        Layout<Shape<Int<M>, Int<K>>, Stride<Int<K>, _1>>,
        Layout<Shape<Int<K>, Int<N>>, Stride<Int<N>, _1>>,
        Layout<Shape<Int<M>, Int<N>>, Stride<Int<N>, _1>>,
        BLOCK_TILE_SIZE_X,
        BLOCK_TILE_SIZE_Y,
        BLOCK_TILE_SIZE_K,
        WARP_TILE_SIZE_X,
        WARP_TILE_SIZE_Y,
        THREAD_TILE_SIZE_X,
        THREAD_TILE_SIZE_Y,
        NUM_THREADS_PER_WARP_X,
        NUM_THREADS_PER_WARP_Y
        ><<<grid_dim, THREADS_PER_BLOCK>>>(
            d_matrixA.data().get(),
            d_matrixB.data().get(),
            d_matrixC.data().get(),
            make_layout(make_shape(Int<M>{}, Int<K>{}), LayoutRight{}),
            make_layout(make_shape(Int<K>{}, Int<N>{}), LayoutRight{}),
            make_layout(make_shape(Int<M>{}, Int<N>{}), LayoutRight{}),
            1.f,
            1.f
        );
}
